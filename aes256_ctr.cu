#include "hip/hip_runtime.h"
#include "aes_common.h"

extern __device__ __constant__ uint32_t d_roundKeys[60];
extern __device__ __constant__ uint32_t d_T0[256], d_T1[256], d_T2[256], d_T3[256];
extern __device__ __constant__ uint8_t  d_sbox[256];

#define CTR_ROUND(o0,o1,o2,o3,s0,s1,s2,s3,rk)                                   \
    do {                                                                       \
        o0 = sh_T0[(s0)&0xFF] ^ sh_T1[((s1)>>8)&0xFF] ^                          \
             sh_T2[((s2)>>16)&0xFF] ^ sh_T3[((s3)>>24)&0xFF] ^ (rk)[0];         \
        o1 = sh_T0[(s1)&0xFF] ^ sh_T1[((s2)>>8)&0xFF] ^                          \
             sh_T2[((s3)>>16)&0xFF] ^ sh_T3[((s0)>>24)&0xFF] ^ (rk)[1];         \
        o2 = sh_T0[(s2)&0xFF] ^ sh_T1[((s3)>>8)&0xFF] ^                          \
             sh_T2[((s0)>>16)&0xFF] ^ sh_T3[((s1)>>24)&0xFF] ^ (rk)[2];         \
        o3 = sh_T0[(s3)&0xFF] ^ sh_T1[((s0)>>8)&0xFF] ^                          \
             sh_T2[((s1)>>16)&0xFF] ^ sh_T3[((s2)>>24)&0xFF] ^ (rk)[3];         \
    } while (0)

__global__ void aes256_ctr_encrypt(const uint8_t *in, uint8_t *out,
                                   size_t nBlocks, uint64_t ctrLo, uint64_t ctrHi) {
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t stride = blockDim.x * gridDim.x;
    if (idx >= nBlocks) return;
    __shared__ uint32_t sh_T0[256], sh_T1[256], sh_T2[256], sh_T3[256];
    __shared__ uint8_t  sh_sbox[256];
    if (threadIdx.x < 256) {
        sh_T0[threadIdx.x] = d_T0[threadIdx.x];
        sh_T1[threadIdx.x] = d_T1[threadIdx.x];
        sh_T2[threadIdx.x] = d_T2[threadIdx.x];
        sh_T3[threadIdx.x] = d_T3[threadIdx.x];
        sh_sbox[threadIdx.x] = d_sbox[threadIdx.x];
    }
    __syncthreads();
    const uint32_t *rk = d_roundKeys;  // 60 words for AES-256

    // -------- first counter --------
    uint64_t ctr_lo = ctrLo + idx;
    uint64_t ctr_hi = ctrHi;
    if (ctr_lo < ctrLo) ctr_hi += 1ULL;

    uint32_t s0 = (uint32_t)ctr_lo;
    uint32_t s1 = (uint32_t)(ctr_lo >> 32);
    uint32_t s2 = (uint32_t)ctr_hi;
    uint32_t s3 = (uint32_t)(ctr_hi >> 32);

    s0 ^= rk[0]; s1 ^= rk[1]; s2 ^= rk[2]; s3 ^= rk[3];
    uint32_t t0, t1, t2, t3;
    CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk + 4);
    s0=t0; s1=t1; s2=t2; s3=t3;
    CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk + 8);
    s0=t0; s1=t1; s2=t2; s3=t3;
    CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +12);
    s0=t0; s1=t1; s2=t2; s3=t3;
    CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +16);
    s0=t0; s1=t1; s2=t2; s3=t3;
    CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +20);
    s0=t0; s1=t1; s2=t2; s3=t3;
    CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +24);
    s0=t0; s1=t1; s2=t2; s3=t3;
    CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +28);
    s0=t0; s1=t1; s2=t2; s3=t3;
    CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +32);
    s0=t0; s1=t1; s2=t2; s3=t3;
    CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +36);
    s0=t0; s1=t1; s2=t2; s3=t3;
    CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +40);
    s0=t0; s1=t1; s2=t2; s3=t3;
    CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +44);
    s0=t0; s1=t1; s2=t2; s3=t3;
    CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +48);
    s0=t0; s1=t1; s2=t2; s3=t3;
    CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +52);
    s0=t0; s1=t1; s2=t2; s3=t3;

    const uint8_t *sb = sh_sbox;
    uint32_t k0 = ((uint32_t)sb[s0 & 0xFF]) |
                  ((uint32_t)sb[s1 & 0xFF] << 8) |
                  ((uint32_t)sb[s2 & 0xFF] << 16) |
                  ((uint32_t)sb[s3 & 0xFF] << 24);
    uint32_t k1 = ((uint32_t)sb[(s1 >> 8) & 0xFF]) |
                  ((uint32_t)sb[(s2 >> 8) & 0xFF] << 8) |
                  ((uint32_t)sb[(s3 >> 8) & 0xFF] << 16) |
                  ((uint32_t)sb[(s0 >> 8) & 0xFF] << 24);
    uint32_t k2 = ((uint32_t)sb[(s2 >> 16) & 0xFF]) |
                  ((uint32_t)sb[(s3 >> 16) & 0xFF] << 8) |
                  ((uint32_t)sb[(s0 >> 16) & 0xFF] << 16) |
                  ((uint32_t)sb[(s1 >> 16) & 0xFF] << 24);
    uint32_t k3 = ((uint32_t)sb[(s3 >> 24) & 0xFF]) |
                  ((uint32_t)sb[(s0 >> 24) & 0xFF] << 8) |
                  ((uint32_t)sb[(s1 >> 24) & 0xFF] << 16) |
                  ((uint32_t)sb[(s2 >> 24) & 0xFF] << 24);

    k0 ^= rk[56]; k1 ^= rk[57]; k2 ^= rk[58]; k3 ^= rk[59];

    uint4 inBlock = reinterpret_cast<const uint4*>(in)[idx];
    uint4 outBlock = make_uint4(inBlock.x ^ k0,
                                inBlock.y ^ k1,
                                inBlock.z ^ k2,
                                inBlock.w ^ k3);
    reinterpret_cast<uint4*>(out)[idx] = outBlock;

    // -------- second counter --------
    size_t idx2 = idx + stride;
    if (idx2 < nBlocks) {
        ctr_lo = ctrLo + idx2;
        ctr_hi = ctrHi;
        if (ctr_lo < ctrLo) ctr_hi += 1ULL;

        s0 = (uint32_t)ctr_lo;
        s1 = (uint32_t)(ctr_lo >> 32);
        s2 = (uint32_t)ctr_hi;
        s3 = (uint32_t)(ctr_hi >> 32);

        s0 ^= rk[0]; s1 ^= rk[1]; s2 ^= rk[2]; s3 ^= rk[3];
        CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk + 4);
        s0=t0; s1=t1; s2=t2; s3=t3;
        CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk + 8);
        s0=t0; s1=t1; s2=t2; s3=t3;
        CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +12);
        s0=t0; s1=t1; s2=t2; s3=t3;
        CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +16);
        s0=t0; s1=t1; s2=t2; s3=t3;
        CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +20);
        s0=t0; s1=t1; s2=t2; s3=t3;
        CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +24);
        s0=t0; s1=t1; s2=t2; s3=t3;
        CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +28);
        s0=t0; s1=t1; s2=t2; s3=t3;
        CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +32);
        s0=t0; s1=t1; s2=t2; s3=t3;
        CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +36);
        s0=t0; s1=t1; s2=t2; s3=t3;
        CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +40);
        s0=t0; s1=t1; s2=t2; s3=t3;
        CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +44);
        s0=t0; s1=t1; s2=t2; s3=t3;
        CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +48);
        s0=t0; s1=t1; s2=t2; s3=t3;
        CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +52);
        s0=t0; s1=t1; s2=t2; s3=t3;

        k0 = ((uint32_t)sb[s0 & 0xFF]) |
             ((uint32_t)sb[s1 & 0xFF] << 8) |
             ((uint32_t)sb[s2 & 0xFF] << 16) |
             ((uint32_t)sb[s3 & 0xFF] << 24);
        k1 = ((uint32_t)sb[(s1 >> 8) & 0xFF]) |
             ((uint32_t)sb[(s2 >> 8) & 0xFF] << 8) |
             ((uint32_t)sb[(s3 >> 8) & 0xFF] << 16) |
             ((uint32_t)sb[(s0 >> 8) & 0xFF] << 24);
        k2 = ((uint32_t)sb[(s2 >> 16) & 0xFF]) |
             ((uint32_t)sb[(s3 >> 16) & 0xFF] << 8) |
             ((uint32_t)sb[(s0 >> 16) & 0xFF] << 16) |
             ((uint32_t)sb[(s1 >> 16) & 0xFF] << 24);
        k3 = ((uint32_t)sb[(s3 >> 24) & 0xFF]) |
             ((uint32_t)sb[(s0 >> 24) & 0xFF] << 8) |
             ((uint32_t)sb[(s1 >> 24) & 0xFF] << 16) |
             ((uint32_t)sb[(s2 >> 24) & 0xFF] << 24);

        k0 ^= rk[56]; k1 ^= rk[57]; k2 ^= rk[58]; k3 ^= rk[59];

        inBlock = reinterpret_cast<const uint4*>(in)[idx2];
        outBlock = make_uint4(inBlock.x ^ k0,
                              inBlock.y ^ k1,
                              inBlock.z ^ k2,
                              inBlock.w ^ k3);
        reinterpret_cast<uint4*>(out)[idx2] = outBlock;
    }
}

__global__ void aes256_ctr_decrypt(const uint8_t *in, uint8_t *out,
                                   size_t nBlocks, uint64_t ctrLo, uint64_t ctrHi) {
    // CTR decryption mirrors encryption; generate the same keystream and XOR.

    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t stride = blockDim.x * gridDim.x;
    if (idx >= nBlocks) return;
    __shared__ uint32_t sh_T0[256], sh_T1[256], sh_T2[256], sh_T3[256];
    __shared__ uint8_t  sh_sbox[256];
    if (threadIdx.x < 256) {
        sh_T0[threadIdx.x] = d_T0[threadIdx.x];
        sh_T1[threadIdx.x] = d_T1[threadIdx.x];
        sh_T2[threadIdx.x] = d_T2[threadIdx.x];
        sh_T3[threadIdx.x] = d_T3[threadIdx.x];
        sh_sbox[threadIdx.x] = d_sbox[threadIdx.x];
    }
    __syncthreads();
    const uint32_t *rk = d_roundKeys;  // 60 words for AES-256

    uint64_t ctr_lo = ctrLo + idx;
    uint64_t ctr_hi = ctrHi;
    if (ctr_lo < ctrLo) ctr_hi += 1ULL;

    uint32_t s0 = (uint32_t)ctr_lo;
    uint32_t s1 = (uint32_t)(ctr_lo >> 32);
    uint32_t s2 = (uint32_t)ctr_hi;
    uint32_t s3 = (uint32_t)(ctr_hi >> 32);

    s0 ^= rk[0]; s1 ^= rk[1]; s2 ^= rk[2]; s3 ^= rk[3];
    uint32_t t0, t1, t2, t3;
    CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk + 4);
    s0=t0; s1=t1; s2=t2; s3=t3;
    CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk + 8);
    s0=t0; s1=t1; s2=t2; s3=t3;
    CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +12);
    s0=t0; s1=t1; s2=t2; s3=t3;
    CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +16);
    s0=t0; s1=t1; s2=t2; s3=t3;
    CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +20);
    s0=t0; s1=t1; s2=t2; s3=t3;
    CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +24);
    s0=t0; s1=t1; s2=t2; s3=t3;
    CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +28);
    s0=t0; s1=t1; s2=t2; s3=t3;
    CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +32);
    s0=t0; s1=t1; s2=t2; s3=t3;
    CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +36);
    s0=t0; s1=t1; s2=t2; s3=t3;
    CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +40);
    s0=t0; s1=t1; s2=t2; s3=t3;
    CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +44);
    s0=t0; s1=t1; s2=t2; s3=t3;
    CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +48);
    s0=t0; s1=t1; s2=t2; s3=t3;
    CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +52);
    s0=t0; s1=t1; s2=t2; s3=t3;

    const uint8_t *sb = sh_sbox;
    uint32_t k0 = ((uint32_t)sb[s0 & 0xFF]) |
                  ((uint32_t)sb[s1 & 0xFF] << 8) |
                  ((uint32_t)sb[s2 & 0xFF] << 16) |
                  ((uint32_t)sb[s3 & 0xFF] << 24);
    uint32_t k1 = ((uint32_t)sb[(s1 >> 8) & 0xFF]) |
                  ((uint32_t)sb[(s2 >> 8) & 0xFF] << 8) |
                  ((uint32_t)sb[(s3 >> 8) & 0xFF] << 16) |
                  ((uint32_t)sb[(s0 >> 8) & 0xFF] << 24);
    uint32_t k2 = ((uint32_t)sb[(s2 >> 16) & 0xFF]) |
                  ((uint32_t)sb[(s3 >> 16) & 0xFF] << 8) |
                  ((uint32_t)sb[(s0 >> 16) & 0xFF] << 16) |
                  ((uint32_t)sb[(s1 >> 16) & 0xFF] << 24);
    uint32_t k3 = ((uint32_t)sb[(s3 >> 24) & 0xFF]) |
                  ((uint32_t)sb[(s0 >> 24) & 0xFF] << 8) |
                  ((uint32_t)sb[(s1 >> 24) & 0xFF] << 16) |
                  ((uint32_t)sb[(s2 >> 24) & 0xFF] << 24);

    k0 ^= rk[56]; k1 ^= rk[57]; k2 ^= rk[58]; k3 ^= rk[59];

    uint4 ctBlock = reinterpret_cast<const uint4*>(in)[idx];
    uint4 ptBlock = make_uint4(ctBlock.x ^ k0,
                               ctBlock.y ^ k1,
                               ctBlock.z ^ k2,
                               ctBlock.w ^ k3);
    reinterpret_cast<uint4*>(out)[idx] = ptBlock;

    size_t idx2 = idx + stride;
    if (idx2 < nBlocks) {
        ctr_lo = ctrLo + idx2;
        ctr_hi = ctrHi;
        if (ctr_lo < ctrLo) ctr_hi += 1ULL;

        s0 = (uint32_t)ctr_lo;
        s1 = (uint32_t)(ctr_lo >> 32);
        s2 = (uint32_t)ctr_hi;
        s3 = (uint32_t)(ctr_hi >> 32);

        s0 ^= rk[0]; s1 ^= rk[1]; s2 ^= rk[2]; s3 ^= rk[3];
        CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk + 4);
        s0=t0; s1=t1; s2=t2; s3=t3;
        CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk + 8);
        s0=t0; s1=t1; s2=t2; s3=t3;
        CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +12);
        s0=t0; s1=t1; s2=t2; s3=t3;
        CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +16);
        s0=t0; s1=t1; s2=t2; s3=t3;
        CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +20);
        s0=t0; s1=t1; s2=t2; s3=t3;
        CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +24);
        s0=t0; s1=t1; s2=t2; s3=t3;
        CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +28);
        s0=t0; s1=t1; s2=t2; s3=t3;
        CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +32);
        s0=t0; s1=t1; s2=t2; s3=t3;
        CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +36);
        s0=t0; s1=t1; s2=t2; s3=t3;
        CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +40);
        s0=t0; s1=t1; s2=t2; s3=t3;
        CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +44);
        s0=t0; s1=t1; s2=t2; s3=t3;
        CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +48);
        s0=t0; s1=t1; s2=t2; s3=t3;
        CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk +52);
        s0=t0; s1=t1; s2=t2; s3=t3;

        k0 = ((uint32_t)sb[s0 & 0xFF]) |
             ((uint32_t)sb[s1 & 0xFF] << 8) |
             ((uint32_t)sb[s2 & 0xFF] << 16) |
             ((uint32_t)sb[s3 & 0xFF] << 24);
        k1 = ((uint32_t)sb[(s1 >> 8) & 0xFF]) |
             ((uint32_t)sb[(s2 >> 8) & 0xFF] << 8) |
             ((uint32_t)sb[(s3 >> 8) & 0xFF] << 16) |
             ((uint32_t)sb[(s0 >> 8) & 0xFF] << 24);
        k2 = ((uint32_t)sb[(s2 >> 16) & 0xFF]) |
             ((uint32_t)sb[(s3 >> 16) & 0xFF] << 8) |
             ((uint32_t)sb[(s0 >> 16) & 0xFF] << 16) |
             ((uint32_t)sb[(s1 >> 16) & 0xFF] << 24);
        k3 = ((uint32_t)sb[(s3 >> 24) & 0xFF]) |
             ((uint32_t)sb[(s0 >> 24) & 0xFF] << 8) |
             ((uint32_t)sb[(s1 >> 24) & 0xFF] << 16) |
             ((uint32_t)sb[(s2 >> 24) & 0xFF] << 24);

        k0 ^= rk[56]; k1 ^= rk[57]; k2 ^= rk[58]; k3 ^= rk[59];

        ctBlock = reinterpret_cast<const uint4*>(in)[idx2];
        ptBlock = make_uint4(ctBlock.x ^ k0,
                             ctBlock.y ^ k1,
                             ctBlock.z ^ k2,
                             ctBlock.w ^ k3);
reinterpret_cast<uint4*>(out)[idx2] = ptBlock;
    }
}

#undef CTR_ROUND
