#include "hip/hip_runtime.h"
#include "aes_common.h"
#include <stdint.h>

extern __device__ __constant__ uint32_t d_roundKeys[60];
extern __device__ __constant__ uint32_t d_T0[256], d_T1[256], d_T2[256], d_T3[256];
extern __device__ __constant__ uint8_t  d_sbox[256];

// Reuse the gf_mul128 device function from aes128_gcm.cu (same implementation)
static __device__ inline void gf_mul128(uint64_t &Ah, uint64_t &Al, uint64_t Bh, uint64_t Bl) {
    uint64_t Zh = 0ull, Zl = 0ull;
    uint64_t Vh = Bh, Vl = Bl;
    const uint64_t R = 0xE100000000000000ULL;
    for (int i = 0; i < 128; ++i) {
        if (Al & 1ULL) {
            Zl ^= Vl; Zh ^= Vh;
        }
        bool carry = (Vl & 1ULL);
        Vl = (Vl >> 1) | (Vh << 63);
        Vh = (Vh >> 1);
        if (carry) Vh ^= R;
        Al = (Al >> 1) | (Ah << 63);
        Ah >>= 1;
    }
    Ah = Zh;
    Al = Zl;
}

__global__ void aes256_gcm_encrypt(const uint8_t *plain, uint8_t *cipher, size_t nBlocks, const uint8_t *iv, uint8_t *tagOut) {
    // Implementation is analogous to aes128_gcm_encrypt, but using AES-256 (14 rounds).
    __shared__ uint64_t sh_H_hi, sh_H_lo;
    __shared__ uint64_t partial_tag_hi[32];
    __shared__ uint64_t partial_tag_lo[32];
    __shared__ uint32_t partial_len[32];

    if (threadIdx.x == 0) {
        // Compute H = AES-256 encrypt of all-zero block
        uint32_t s0=0, s1=0, s2=0, s3=0;
        const uint32_t *rk = d_roundKeys;
        s0 ^= rk[0]; s1 ^= rk[1]; s2 ^= rk[2]; s3 ^= rk[3];
        uint32_t t0,t1,t2,t3;
        #pragma unroll
        for (int r = 1; r <= 13; ++r) {
            t0 = d_T0[s0 & 0xFF] ^ d_T1[(s1>>8)&0xFF] ^ d_T2[(s2>>16)&0xFF] ^ d_T3[(s3>>24)&0xFF] ^ rk[4*r+0];
            t1 = d_T0[s1 & 0xFF] ^ d_T1[(s2>>8)&0xFF] ^ d_T2[(s3>>16)&0xFF] ^ d_T3[(s0>>24)&0xFF] ^ rk[4*r+1];
            t2 = d_T0[s2 & 0xFF] ^ d_T1[(s3>>8)&0xFF] ^ d_T2[(s0>>16)&0xFF] ^ d_T3[(s1>>24)&0xFF] ^ rk[4*r+2];
            t3 = d_T0[s3 & 0xFF] ^ d_T1[(s0>>8)&0xFF] ^ d_T2[(s1>>16)&0xFF] ^ d_T3[(s2>>24)&0xFF] ^ rk[4*r+3];
            s0=t0; s1=t1; s2=t2; s3=t3;
        }
        uint8_t buf[16];
        const uint8_t *sb = d_sbox;
        buf[0]  = sb[s0 & 0xFF];      buf[4]  = sb[(s1>>8)&0xFF];
        buf[8]  = sb[(s2>>16)&0xFF]; buf[12] = sb[(s3>>24)&0xFF];
        buf[1]  = sb[s1 & 0xFF];      buf[5]  = sb[(s2>>8)&0xFF];
        buf[9]  = sb[(s3>>16)&0xFF]; buf[13] = sb[(s0>>24)&0xFF];
        buf[2]  = sb[s2 & 0xFF];      buf[6]  = sb[(s3>>8)&0xFF];
        buf[10] = sb[(s0>>16)&0xFF]; buf[14] = sb[(s1>>24)&0xFF];
        buf[3]  = sb[s3 & 0xFF];      buf[7]  = sb[(s0>>8)&0xFF];
        buf[11] = sb[(s1>>16)&0xFF]; buf[15] = sb[(s2>>24)&0xFF];
        ((uint32_t*)buf)[0] ^= rk[56];
        ((uint32_t*)buf)[1] ^= rk[57];
        ((uint32_t*)buf)[2] ^= rk[58];
        ((uint32_t*)buf)[3] ^= rk[59];
        sh_H_lo = ((uint64_t*)buf)[0];
        sh_H_hi = ((uint64_t*)buf)[1];
    }
    __syncthreads();

    // Setup IV counters
    uint64_t IV_lo = 0ull, IV_hi = 0ull;
    if (threadIdx.x == 0) {
        IV_hi = ((const uint64_t*)iv)[0];
        uint32_t iv_low32 = 0;
        memcpy(&iv_low32, iv + 8, 4);
        IV_lo = ((uint64_t)iv_low32 << 32) | 1ULL;
    }
    __syncthreads();
    IV_lo = __shfl_sync(0xFFFFFFFF, IV_lo, 0);
    IV_hi = __shfl_sync(0xFFFFFFFF, IV_hi, 0);

    // CTR encryption (each thread processes blocks in stride)
    for (size_t i = threadIdx.x; i < nBlocks; i += blockDim.x) {
        uint64_t ctr_lo = IV_lo + i;
        uint64_t ctr_hi = IV_hi;
        if (ctr_lo < IV_lo) ctr_hi += 1ULL;
        uint32_t s0 = (uint32_t)ctr_lo;
        uint32_t s1 = (uint32_t)(ctr_lo >> 32);
        uint32_t s2 = (uint32_t)ctr_hi;
        uint32_t s3 = (uint32_t)(ctr_hi >> 32);
        const uint32_t *rk = d_roundKeys;
        s0 ^= rk[0]; s1 ^= rk[1]; s2 ^= rk[2]; s3 ^= rk[3];
        uint32_t t0,t1,t2,t3;
        #pragma unroll
        for (int r = 1; r <= 13; ++r) {
            t0 = d_T0[s0 & 0xFF] ^ d_T1[(s1>>8)&0xFF] ^ d_T2[(s2>>16)&0xFF] ^ d_T3[(s3>>24)&0xFF] ^ rk[4*r+0];
            t1 = d_T0[s1 & 0xFF] ^ d_T1[(s2>>8)&0xFF] ^ d_T2[(s3>>16)&0xFF] ^ d_T3[(s0>>24)&0xFF] ^ rk[4*r+1];
            t2 = d_T0[s2 & 0xFF] ^ d_T1[(s3>>8)&0xFF] ^ d_T2[(s0>>16)&0xFF] ^ d_T3[(s1>>24)&0xFF] ^ rk[4*r+2];
            t3 = d_T0[s3 & 0xFF] ^ d_T1[(s0>>8)&0xFF] ^ d_T2[(s1>>16)&0xFF] ^ d_T3[(s2>>24)&0xFF] ^ rk[4*r+3];
            s0=t0; s1=t1; s2=t2; s3=t3;
        }
        uint8_t keystream[16];
        uint8_t *ko = keystream;
        const uint8_t *sb = d_sbox;
        ko[0]  = sb[s0 & 0xFF];       ko[4]  = sb[(s1>>8)&0xFF];
        ko[8]  = sb[(s2>>16)&0xFF];   ko[12] = sb[(s3>>24)&0xFF];
        ko[1]  = sb[s1 & 0xFF];       ko[5]  = sb[(s2>>8)&0xFF];
        ko[9]  = sb[(s3>>16)&0xFF];   ko[13] = sb[(s0>>24)&0xFF];
        ko[2]  = sb[s2 & 0xFF];       ko[6]  = sb[(s3>>8)&0xFF];
        ko[10] = sb[(s0>>16)&0xFF];   ko[14] = sb[(s1>>24)&0xFF];
        ko[3]  = sb[s3 & 0xFF];       ko[7]  = sb[(s0>>8)&0xFF];
        ko[11] = sb[(s1>>16)&0xFF];   ko[15] = sb[(s2>>24)&0xFF];
        ((uint32_t*)keystream)[0] ^= rk[56];
        ((uint32_t*)keystream)[1] ^= rk[57];
        ((uint32_t*)keystream)[2] ^= rk[58];
        ((uint32_t*)keystream)[3] ^= rk[59];
        const uint8_t *ct_in = plain + i * 16;
        uint8_t *ct_out = cipher + i * 16;
        ((uint32_t*)ct_out)[0] = ((const uint32_t*)ct_in)[0] ^ ((uint32_t*)keystream)[0];
        ((uint32_t*)ct_out)[1] = ((const uint32_t*)ct_in)[1] ^ ((uint32_t*)keystream)[1];
        ((uint32_t*)ct_out)[2] = ((const uint32_t*)ct_in)[2] ^ ((uint32_t*)keystream)[2];
        ((uint32_t*)ct_out)[3] = ((const uint32_t*)ct_in)[3] ^ ((uint32_t*)keystream)[3];
    }
    __syncthreads();

    // GHASH (same pattern as AES-128 GCM, with same polynomial multiply)
    uint32_t tid = threadIdx.x;
    if (tid < 32) {
        size_t seg_size = (nBlocks + 31) / 32;
        size_t start = seg_size * tid;
        size_t end = (start + seg_size < nBlocks) ? (start + seg_size) : nBlocks;
        uint64_t Xh = 0ull, Xl = 0ull;
        for (size_t j = start; j < end; ++j) {
            uint64_t c_l = ((uint64_t*)cipher)[2*j + 0];
            uint64_t c_h = ((uint64_t*)cipher)[2*j + 1];
            Xl ^= c_l;
            Xh ^= c_h;
            gf_mul128(Xh, Xl, sh_H_hi, sh_H_lo);
        }
        partial_tag_hi[tid] = Xh;
        partial_tag_lo[tid] = Xl;
        partial_len[tid] = (uint32_t)(end - start);
    }
    __syncthreads();
    if (threadIdx.x == 0) {
        uint64_t tag_h = partial_tag_hi[0];
        uint64_t tag_l = partial_tag_lo[0];
        uint64_t pow_hi[27], pow_lo[27];
        pow_hi[0] = sh_H_hi;
        pow_lo[0] = sh_H_lo;
        for (int i = 1; i < 27; ++i) {
            pow_hi[i] = pow_hi[i-1];
            pow_lo[i] = pow_lo[i-1];
            gf_mul128(pow_hi[i], pow_lo[i], pow_hi[i-1], pow_lo[i-1]);
        }
        size_t done_blocks = partial_len[0];
        for (int seg = 1; seg < 32; ++seg) {
            uint64_t exp_hi = 0ull, exp_lo = 0ull;
            exp_lo = 1ull; exp_hi = 0ull;
            uint32_t count = partial_len[seg];
            for (int b = 0; b < 27; ++b) {
                if (count & (1u << b)) {
                    gf_mul128(exp_hi, exp_lo, pow_hi[b], pow_lo[b]);
                }
            }
            gf_mul128(tag_h, tag_l, exp_hi, exp_lo);
            tag_h ^= partial_tag_hi[seg];
            tag_l ^= partial_tag_lo[seg];
            done_blocks += partial_len[seg];
        }
        ((uint64_t*)tagOut)[0] = tag_l;
        ((uint64_t*)tagOut)[1] = tag_h;
    }
}

__global__ void aes256_gcm_decrypt(const uint8_t *cipher, uint8_t *plain, size_t nBlocks, const uint8_t *iv, const uint8_t *tag, uint8_t *tagOut) {
    // Mirror aes256_gcm_encrypt but with ciphertext as input and plaintext as
    // output.  The authentication tag for the ciphertext is produced in
    // tagOut; the provided 'tag' parameter is expected to be checked by the
    // host.

    __shared__ uint64_t sh_H_hi, sh_H_lo;
    __shared__ uint64_t partial_tag_hi[32];
    __shared__ uint64_t partial_tag_lo[32];
    __shared__ uint32_t partial_len[32];

    if (threadIdx.x == 0) {
        uint32_t s0=0, s1=0, s2=0, s3=0;
        const uint32_t *rk = d_roundKeys;
        s0 ^= rk[0]; s1 ^= rk[1]; s2 ^= rk[2]; s3 ^= rk[3];
        uint32_t t0,t1,t2,t3;
#pragma unroll
        for (int r = 1; r <= 13; ++r) {
            t0 = d_T0[s0 & 0xFF] ^ d_T1[(s1>>8)&0xFF] ^ d_T2[(s2>>16)&0xFF] ^ d_T3[(s3>>24)&0xFF] ^ rk[4*r+0];
            t1 = d_T0[s1 & 0xFF] ^ d_T1[(s2>>8)&0xFF] ^ d_T2[(s3>>16)&0xFF] ^ d_T3[(s0>>24)&0xFF] ^ rk[4*r+1];
            t2 = d_T0[s2 & 0xFF] ^ d_T1[(s3>>8)&0xFF] ^ d_T2[(s0>>16)&0xFF] ^ d_T3[(s1>>24)&0xFF] ^ rk[4*r+2];
            t3 = d_T0[s3 & 0xFF] ^ d_T1[(s0>>8)&0xFF] ^ d_T2[(s1>>16)&0xFF] ^ d_T3[(s2>>24)&0xFF] ^ rk[4*r+3];
            s0=t0; s1=t1; s2=t2; s3=t3;
        }
        uint8_t buf[16];
        const uint8_t *sb = d_sbox;
        buf[0]  = sb[s0 & 0xFF];      buf[4]  = sb[(s1>>8)&0xFF];
        buf[8]  = sb[(s2>>16)&0xFF];  buf[12] = sb[(s3>>24)&0xFF];
        buf[1]  = sb[s1 & 0xFF];      buf[5]  = sb[(s2>>8)&0xFF];
        buf[9]  = sb[(s3>>16)&0xFF];  buf[13] = sb[(s0>>24)&0xFF];
        buf[2]  = sb[s2 & 0xFF];      buf[6]  = sb[(s3>>8)&0xFF];
        buf[10] = sb[(s0>>16)&0xFF];  buf[14] = sb[(s1>>24)&0xFF];
        buf[3]  = sb[s3 & 0xFF];      buf[7]  = sb[(s0>>8)&0xFF];
        buf[11] = sb[(s1>>16)&0xFF];  buf[15] = sb[(s2>>24)&0xFF];
        ((uint32_t*)buf)[0] ^= rk[56];
        ((uint32_t*)buf)[1] ^= rk[57];
        ((uint32_t*)buf)[2] ^= rk[58];
        ((uint32_t*)buf)[3] ^= rk[59];
        sh_H_lo = ((uint64_t*)buf)[0];
        sh_H_hi = ((uint64_t*)buf)[1];
    }
    __syncthreads();

    uint64_t IV_lo = 0ull, IV_hi = 0ull;
    if (threadIdx.x == 0) {
        IV_hi = ((const uint64_t*)iv)[0];
        uint32_t iv_low32 = 0;
        memcpy(&iv_low32, iv + 8, 4);
        IV_lo = ((uint64_t)iv_low32 << 32) | 1ULL;
    }
    __syncthreads();
    IV_lo = __shfl_sync(0xFFFFFFFF, IV_lo, 0);
    IV_hi = __shfl_sync(0xFFFFFFFF, IV_hi, 0);

    for (size_t i = threadIdx.x; i < nBlocks; i += blockDim.x) {
        uint64_t ctr_lo = IV_lo + i;
        uint64_t ctr_hi = IV_hi;
        if (ctr_lo < IV_lo) ctr_hi += 1ULL;
        uint32_t s0 = (uint32_t)ctr_lo;
        uint32_t s1 = (uint32_t)(ctr_lo >> 32);
        uint32_t s2 = (uint32_t)ctr_hi;
        uint32_t s3 = (uint32_t)(ctr_hi >> 32);
        const uint32_t *rk = d_roundKeys;
        s0 ^= rk[0]; s1 ^= rk[1]; s2 ^= rk[2]; s3 ^= rk[3];
        uint32_t t0,t1,t2,t3;
#pragma unroll
        for (int r = 1; r <= 13; ++r) {
            t0 = d_T0[s0 & 0xFF] ^ d_T1[(s1>>8)&0xFF] ^ d_T2[(s2>>16)&0xFF] ^ d_T3[(s3>>24)&0xFF] ^ rk[4*r+0];
            t1 = d_T0[s1 & 0xFF] ^ d_T1[(s2>>8)&0xFF] ^ d_T2[(s3>>16)&0xFF] ^ d_T3[(s0>>24)&0xFF] ^ rk[4*r+1];
            t2 = d_T0[s2 & 0xFF] ^ d_T1[(s3>>8)&0xFF] ^ d_T2[(s0>>16)&0xFF] ^ d_T3[(s1>>24)&0xFF] ^ rk[4*r+2];
            t3 = d_T0[s3 & 0xFF] ^ d_T1[(s0>>8)&0xFF] ^ d_T2[(s1>>16)&0xFF] ^ d_T3[(s2>>24)&0xFF] ^ rk[4*r+3];
            s0=t0; s1=t1; s2=t2; s3=t3;
        }
        uint8_t keystream[16];
        uint8_t *ko = keystream;
        const uint8_t *sb = d_sbox;
        ko[0]  = sb[s0 & 0xFF];       ko[4]  = sb[(s1>>8)&0xFF];
        ko[8]  = sb[(s2>>16)&0xFF];   ko[12] = sb[(s3>>24)&0xFF];
        ko[1]  = sb[s1 & 0xFF];       ko[5]  = sb[(s2>>8)&0xFF];
        ko[9]  = sb[(s3>>16)&0xFF];   ko[13] = sb[(s0>>24)&0xFF];
        ko[2]  = sb[s2 & 0xFF];       ko[6]  = sb[(s3>>8)&0xFF];
        ko[10] = sb[(s0>>16)&0xFF];   ko[14] = sb[(s1>>24)&0xFF];
        ko[3]  = sb[s3 & 0xFF];       ko[7]  = sb[(s0>>8)&0xFF];
        ko[11] = sb[(s1>>16)&0xFF];   ko[15] = sb[(s2>>24)&0xFF];
        ((uint32_t*)keystream)[0] ^= rk[56];
        ((uint32_t*)keystream)[1] ^= rk[57];
        ((uint32_t*)keystream)[2] ^= rk[58];
        ((uint32_t*)keystream)[3] ^= rk[59];
        const uint8_t *ct_block = cipher + i * 16;
        uint8_t *pt_block = plain + i * 16;
        ((uint32_t*)pt_block)[0] = ((const uint32_t*)ct_block)[0] ^ ((uint32_t*)keystream)[0];
        ((uint32_t*)pt_block)[1] = ((const uint32_t*)ct_block)[1] ^ ((uint32_t*)keystream)[1];
        ((uint32_t*)pt_block)[2] = ((const uint32_t*)ct_block)[2] ^ ((uint32_t*)keystream)[2];
        ((uint32_t*)pt_block)[3] = ((const uint32_t*)ct_block)[3] ^ ((uint32_t*)keystream)[3];
    }
    __syncthreads();

    uint32_t tid = threadIdx.x;
    if (tid < 32) {
        size_t seg_size = (nBlocks + 31) / 32;
        size_t start = seg_size * tid;
        size_t end = (start + seg_size < nBlocks) ? (start + seg_size) : nBlocks;
        uint64_t Xh = 0ull, Xl = 0ull;
        for (size_t j = start; j < end; ++j) {
            uint64_t c_l = ((uint64_t*)cipher)[2*j + 0];
            uint64_t c_h = ((uint64_t*)cipher)[2*j + 1];
            Xl ^= c_l;
            Xh ^= c_h;
            gf_mul128(Xh, Xl, sh_H_hi, sh_H_lo);
        }
        partial_tag_hi[tid] = Xh;
        partial_tag_lo[tid] = Xl;
        partial_len[tid] = (uint32_t)(end - start);
    }
    __syncthreads();
    if (threadIdx.x == 0) {
        uint64_t tag_h = partial_tag_hi[0];
        uint64_t tag_l = partial_tag_lo[0];
        uint64_t pow_hi[27], pow_lo[27];
        pow_hi[0] = sh_H_hi;
        pow_lo[0] = sh_H_lo;
        for (int i = 1; i < 27; ++i) {
            pow_hi[i] = pow_hi[i-1];
            pow_lo[i] = pow_lo[i-1];
            gf_mul128(pow_hi[i], pow_lo[i], pow_hi[i-1], pow_lo[i-1]);
        }
        size_t done_blocks = partial_len[0];
        for (int seg = 1; seg < 32; ++seg) {
            uint64_t exp_hi = 0ull, exp_lo = 0ull;
            exp_lo = 1ull; exp_hi = 0ull;
            uint32_t count = partial_len[seg];
            for (int b = 0; b < 27; ++b) {
                if (count & (1u << b)) {
                    gf_mul128(exp_hi, exp_lo, pow_hi[b], pow_lo[b]);
                }
            }
            gf_mul128(tag_h, tag_l, exp_hi, exp_lo);
            tag_h ^= partial_tag_hi[seg];
            tag_l ^= partial_tag_lo[seg];
            done_blocks += partial_len[seg];
        }
        ((uint64_t*)tagOut)[0] = tag_l;
        ((uint64_t*)tagOut)[1] = tag_h;
    }
}
