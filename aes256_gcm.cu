#include "hip/hip_runtime.h"
#include "aes_common.h"
#include <stdint.h>

extern __device__ __constant__ uint32_t d_roundKeys[60];
extern __device__ __constant__ uint32_t d_T0[256], d_T1[256], d_T2[256], d_T3[256];
extern __device__ __constant__ uint8_t  d_sbox[256];

// Maximum exponent bits used for powers of H (supports up to 2^27 blocks)
static const int MAX_POW_BITS = 27;

// Reuse the gf_mul128 device function from aes128_gcm.cu (same implementation)
static __device__ inline void gf_mul128(uint64_t &Ah, uint64_t &Al,
                                        uint64_t Bh, uint64_t Bl) {
    uint64_t Zh = 0ull, Zl = 0ull;
    uint64_t Vh = Bh, Vl = Bl;
    const uint64_t R = 0xE100000000000000ULL;
    for (int i = 0; i < 128; ++i) {
        if (Al & 1ULL) {
            Zl ^= Vl; Zh ^= Vh;
        }
        bool carry = (Vl & 1ULL);
        Vl = (Vl >> 1) | (Vh << 63);
        Vh >>= 1;
        if (carry) Vh ^= R;
        Al = (Al >> 1) | (Ah << 63);
        Ah >>= 1;
    }
    Ah = Zh;
    Al = Zl;
}

__global__ void aes256_gcm_encrypt(const uint8_t *plain, uint8_t *cipher, size_t nBlocks, const uint8_t *iv, uint8_t *tagOut) {
    // Implementation is analogous to aes128_gcm_encrypt, but using AES-256 (14 rounds).
    __shared__ uint64_t sh_H_hi, sh_H_lo;
    // Shared memory for GHASH reduction and powers of H
    __shared__ uint64_t partial_tag_hi[256];
    __shared__ uint64_t partial_tag_lo[256];
    __shared__ uint64_t pow_H_hi[MAX_POW_BITS];
    __shared__ uint64_t pow_H_lo[MAX_POW_BITS];

    if (threadIdx.x == 0) {
        // Compute H = AES-256 encrypt of all-zero block
        uint32_t s0=0, s1=0, s2=0, s3=0;
        const uint32_t *rk = d_roundKeys;
        s0 ^= rk[0]; s1 ^= rk[1]; s2 ^= rk[2]; s3 ^= rk[3];
        uint32_t t0,t1,t2,t3;
        #pragma unroll
        for (int r = 1; r <= 13; ++r) {
            t0 = d_T0[s0 & 0xFF] ^ d_T1[(s1>>8)&0xFF] ^ d_T2[(s2>>16)&0xFF] ^ d_T3[(s3>>24)&0xFF] ^ rk[4*r+0];
            t1 = d_T0[s1 & 0xFF] ^ d_T1[(s2>>8)&0xFF] ^ d_T2[(s3>>16)&0xFF] ^ d_T3[(s0>>24)&0xFF] ^ rk[4*r+1];
            t2 = d_T0[s2 & 0xFF] ^ d_T1[(s3>>8)&0xFF] ^ d_T2[(s0>>16)&0xFF] ^ d_T3[(s1>>24)&0xFF] ^ rk[4*r+2];
            t3 = d_T0[s3 & 0xFF] ^ d_T1[(s0>>8)&0xFF] ^ d_T2[(s1>>16)&0xFF] ^ d_T3[(s2>>24)&0xFF] ^ rk[4*r+3];
            s0=t0; s1=t1; s2=t2; s3=t3;
        }
        uint8_t buf[16];
        const uint8_t *sb = d_sbox;
        buf[0]  = sb[s0 & 0xFF];      buf[4]  = sb[(s1>>8)&0xFF];
        buf[8]  = sb[(s2>>16)&0xFF]; buf[12] = sb[(s3>>24)&0xFF];
        buf[1]  = sb[s1 & 0xFF];      buf[5]  = sb[(s2>>8)&0xFF];
        buf[9]  = sb[(s3>>16)&0xFF]; buf[13] = sb[(s0>>24)&0xFF];
        buf[2]  = sb[s2 & 0xFF];      buf[6]  = sb[(s3>>8)&0xFF];
        buf[10] = sb[(s0>>16)&0xFF]; buf[14] = sb[(s1>>24)&0xFF];
        buf[3]  = sb[s3 & 0xFF];      buf[7]  = sb[(s0>>8)&0xFF];
        buf[11] = sb[(s1>>16)&0xFF]; buf[15] = sb[(s2>>24)&0xFF];
        ((uint32_t*)buf)[0] ^= rk[56];
        ((uint32_t*)buf)[1] ^= rk[57];
        ((uint32_t*)buf)[2] ^= rk[58];
        ((uint32_t*)buf)[3] ^= rk[59];
        sh_H_lo = ((uint64_t*)buf)[0];
        sh_H_hi = ((uint64_t*)buf)[1];
    }
    __syncthreads();

    for (int k = threadIdx.x; k < MAX_POW_BITS; k += blockDim.x) {
        uint64_t ph=0ull, pl=1ull, bh=sh_H_hi, bl=sh_H_lo; unsigned exp=1u<<k;
        while (exp) {
            if (exp & 1u) gf_mul128(ph, pl, bh, bl);
            exp >>= 1;
            if (exp) gf_mul128(bh, bl, bh, bl);
        }
        pow_H_hi[k]=ph; pow_H_lo[k]=pl;
    }
    __syncthreads();

    // Setup IV counters
    uint64_t IV_lo = 0ull, IV_hi = 0ull;
    if (threadIdx.x == 0) {
        uint32_t w0 = 0, w1 = 0, w2 = 0;
        memcpy(&w0, iv, 4);
        memcpy(&w1, iv + 4, 4);
        memcpy(&w2, iv + 8, 4);
        uint32_t w3 = 0x01000000u;
        IV_lo = (uint64_t)w0 | ((uint64_t)w1 << 32);
        IV_hi = (uint64_t)w2 | ((uint64_t)w3 << 32);
    }
    __syncthreads();
    IV_lo = __shfl_sync(0xFFFFFFFF, IV_lo, 0);
    IV_hi = __shfl_sync(0xFFFFFFFF, IV_hi, 0);

    // CTR encryption (each thread processes blocks in stride)
    for (size_t i = threadIdx.x; i < nBlocks; i += blockDim.x) {
        uint64_t ctr_lo = IV_lo + i;
        uint64_t ctr_hi = IV_hi;
        if (ctr_lo < IV_lo) ctr_hi += 1ULL;
        uint32_t s0 = (uint32_t)ctr_lo;
        uint32_t s1 = (uint32_t)(ctr_lo >> 32);
        uint32_t s2 = (uint32_t)ctr_hi;
        uint32_t s3 = (uint32_t)(ctr_hi >> 32);
        const uint32_t *rk = d_roundKeys;
        s0 ^= rk[0]; s1 ^= rk[1]; s2 ^= rk[2]; s3 ^= rk[3];
        uint32_t t0,t1,t2,t3;
        #pragma unroll
        for (int r = 1; r <= 13; ++r) {
            t0 = d_T0[s0 & 0xFF] ^ d_T1[(s1>>8)&0xFF] ^ d_T2[(s2>>16)&0xFF] ^ d_T3[(s3>>24)&0xFF] ^ rk[4*r+0];
            t1 = d_T0[s1 & 0xFF] ^ d_T1[(s2>>8)&0xFF] ^ d_T2[(s3>>16)&0xFF] ^ d_T3[(s0>>24)&0xFF] ^ rk[4*r+1];
            t2 = d_T0[s2 & 0xFF] ^ d_T1[(s3>>8)&0xFF] ^ d_T2[(s0>>16)&0xFF] ^ d_T3[(s1>>24)&0xFF] ^ rk[4*r+2];
            t3 = d_T0[s3 & 0xFF] ^ d_T1[(s0>>8)&0xFF] ^ d_T2[(s1>>16)&0xFF] ^ d_T3[(s2>>24)&0xFF] ^ rk[4*r+3];
            s0=t0; s1=t1; s2=t2; s3=t3;
        }
        uint8_t keystream[16];
        uint8_t *ko = keystream;
        const uint8_t *sb = d_sbox;
        ko[0]  = sb[s0 & 0xFF];       ko[4]  = sb[(s1>>8)&0xFF];
        ko[8]  = sb[(s2>>16)&0xFF];   ko[12] = sb[(s3>>24)&0xFF];
        ko[1]  = sb[s1 & 0xFF];       ko[5]  = sb[(s2>>8)&0xFF];
        ko[9]  = sb[(s3>>16)&0xFF];   ko[13] = sb[(s0>>24)&0xFF];
        ko[2]  = sb[s2 & 0xFF];       ko[6]  = sb[(s3>>8)&0xFF];
        ko[10] = sb[(s0>>16)&0xFF];   ko[14] = sb[(s1>>24)&0xFF];
        ko[3]  = sb[s3 & 0xFF];       ko[7]  = sb[(s0>>8)&0xFF];
        ko[11] = sb[(s1>>16)&0xFF];   ko[15] = sb[(s2>>24)&0xFF];
        ((uint32_t*)keystream)[0] ^= rk[56];
        ((uint32_t*)keystream)[1] ^= rk[57];
        ((uint32_t*)keystream)[2] ^= rk[58];
        ((uint32_t*)keystream)[3] ^= rk[59];
        const uint8_t *ct_in = plain + i * 16;
        uint8_t *ct_out = cipher + i * 16;
        ((uint32_t*)ct_out)[0] = ((const uint32_t*)ct_in)[0] ^ ((uint32_t*)keystream)[0];
        ((uint32_t*)ct_out)[1] = ((const uint32_t*)ct_in)[1] ^ ((uint32_t*)keystream)[1];
        ((uint32_t*)ct_out)[2] = ((const uint32_t*)ct_in)[2] ^ ((uint32_t*)keystream)[2];
        ((uint32_t*)ct_out)[3] = ((const uint32_t*)ct_in)[3] ^ ((uint32_t*)keystream)[3];
    }
    __syncthreads();

    // GHASH (same pattern as AES-128 GCM, with same polynomial multiply)
    uint32_t tid = threadIdx.x;
    uint64_t partial_h = 0ull, partial_l = 0ull;
    for (size_t i = tid; i < nBlocks; i += blockDim.x) {
        uint64_t c_l = ((uint64_t*)cipher)[2*i + 0];
        uint64_t c_h = ((uint64_t*)cipher)[2*i + 1];
        size_t exp = nBlocks - i;
        uint64_t pow_h = 0ull, pow_l = 1ull;
        for (int b=0; b<MAX_POW_BITS; ++b) {
            if (exp & (1u<<b)) {
                gf_mul128(pow_h, pow_l, pow_H_hi[b], pow_H_lo[b]);
            }
        }
        gf_mul128(c_h, c_l, pow_h, pow_l);
        partial_h ^= c_h;
        partial_l ^= c_l;
    }
    partial_tag_hi[tid] = partial_h;
    partial_tag_lo[tid] = partial_l;
    __syncthreads();
    for (unsigned s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            partial_tag_hi[tid] ^= partial_tag_hi[tid + s];
            partial_tag_lo[tid] ^= partial_tag_lo[tid + s];
        }
        __syncthreads();
    }
    if (tid == 0) {
        ((uint64_t*)tagOut)[0] = partial_tag_lo[0];
        ((uint64_t*)tagOut)[1] = partial_tag_hi[0];
    }
}

__global__ void aes256_gcm_decrypt(const uint8_t *cipher, uint8_t *plain, size_t nBlocks, const uint8_t *iv, const uint8_t *tag, uint8_t *tagOut) {
    // Mirror aes256_gcm_encrypt but with ciphertext as input and plaintext as
    // output.  The authentication tag for the ciphertext is produced in
    // tagOut; the provided 'tag' parameter is expected to be checked by the
    // host.

    __shared__ uint64_t sh_H_hi, sh_H_lo;
    __shared__ uint64_t partial_tag_hi[256];
    __shared__ uint64_t partial_tag_lo[256];
    __shared__ uint64_t pow_H_hi[MAX_POW_BITS];
    __shared__ uint64_t pow_H_lo[MAX_POW_BITS];

    if (threadIdx.x == 0) {
        uint32_t s0=0, s1=0, s2=0, s3=0;
        const uint32_t *rk = d_roundKeys;
        s0 ^= rk[0]; s1 ^= rk[1]; s2 ^= rk[2]; s3 ^= rk[3];
        uint32_t t0,t1,t2,t3;
#pragma unroll
        for (int r = 1; r <= 13; ++r) {
            t0 = d_T0[s0 & 0xFF] ^ d_T1[(s1>>8)&0xFF] ^ d_T2[(s2>>16)&0xFF] ^ d_T3[(s3>>24)&0xFF] ^ rk[4*r+0];
            t1 = d_T0[s1 & 0xFF] ^ d_T1[(s2>>8)&0xFF] ^ d_T2[(s3>>16)&0xFF] ^ d_T3[(s0>>24)&0xFF] ^ rk[4*r+1];
            t2 = d_T0[s2 & 0xFF] ^ d_T1[(s3>>8)&0xFF] ^ d_T2[(s0>>16)&0xFF] ^ d_T3[(s1>>24)&0xFF] ^ rk[4*r+2];
            t3 = d_T0[s3 & 0xFF] ^ d_T1[(s0>>8)&0xFF] ^ d_T2[(s1>>16)&0xFF] ^ d_T3[(s2>>24)&0xFF] ^ rk[4*r+3];
            s0=t0; s1=t1; s2=t2; s3=t3;
        }
        uint8_t buf[16];
        const uint8_t *sb = d_sbox;
        buf[0]  = sb[s0 & 0xFF];      buf[4]  = sb[(s1>>8)&0xFF];
        buf[8]  = sb[(s2>>16)&0xFF];  buf[12] = sb[(s3>>24)&0xFF];
        buf[1]  = sb[s1 & 0xFF];      buf[5]  = sb[(s2>>8)&0xFF];
        buf[9]  = sb[(s3>>16)&0xFF];  buf[13] = sb[(s0>>24)&0xFF];
        buf[2]  = sb[s2 & 0xFF];      buf[6]  = sb[(s3>>8)&0xFF];
        buf[10] = sb[(s0>>16)&0xFF];  buf[14] = sb[(s1>>24)&0xFF];
        buf[3]  = sb[s3 & 0xFF];      buf[7]  = sb[(s0>>8)&0xFF];
        buf[11] = sb[(s1>>16)&0xFF];  buf[15] = sb[(s2>>24)&0xFF];
        ((uint32_t*)buf)[0] ^= rk[56];
        ((uint32_t*)buf)[1] ^= rk[57];
        ((uint32_t*)buf)[2] ^= rk[58];
        ((uint32_t*)buf)[3] ^= rk[59];
        sh_H_lo = ((uint64_t*)buf)[0];
        sh_H_hi = ((uint64_t*)buf)[1];
    }
    __syncthreads();

    for (int k = threadIdx.x; k < MAX_POW_BITS; k += blockDim.x) {
        uint64_t ph=0ull, pl=1ull, bh=sh_H_hi, bl=sh_H_lo; unsigned exp=1u<<k;
        while (exp) {
            if (exp & 1u) gf_mul128(ph, pl, bh, bl);
            exp >>= 1;
            if (exp) gf_mul128(bh, bl, bh, bl);
        }
        pow_H_hi[k]=ph; pow_H_lo[k]=pl;
    }
    __syncthreads();

    uint64_t IV_lo = 0ull, IV_hi = 0ull;
    if (threadIdx.x == 0) {
        uint32_t w0 = 0, w1 = 0, w2 = 0;
        memcpy(&w0, iv, 4);
        memcpy(&w1, iv + 4, 4);
        memcpy(&w2, iv + 8, 4);
        uint32_t w3 = 0x01000000u;
        IV_lo = (uint64_t)w0 | ((uint64_t)w1 << 32);
        IV_hi = (uint64_t)w2 | ((uint64_t)w3 << 32);
    }
    __syncthreads();
    IV_lo = __shfl_sync(0xFFFFFFFF, IV_lo, 0);
    IV_hi = __shfl_sync(0xFFFFFFFF, IV_hi, 0);

    for (size_t i = threadIdx.x; i < nBlocks; i += blockDim.x) {
        uint64_t ctr_lo = IV_lo + i;
        uint64_t ctr_hi = IV_hi;
        if (ctr_lo < IV_lo) ctr_hi += 1ULL;
        uint32_t s0 = (uint32_t)ctr_lo;
        uint32_t s1 = (uint32_t)(ctr_lo >> 32);
        uint32_t s2 = (uint32_t)ctr_hi;
        uint32_t s3 = (uint32_t)(ctr_hi >> 32);
        const uint32_t *rk = d_roundKeys;
        s0 ^= rk[0]; s1 ^= rk[1]; s2 ^= rk[2]; s3 ^= rk[3];
        uint32_t t0,t1,t2,t3;
#pragma unroll
        for (int r = 1; r <= 13; ++r) {
            t0 = d_T0[s0 & 0xFF] ^ d_T1[(s1>>8)&0xFF] ^ d_T2[(s2>>16)&0xFF] ^ d_T3[(s3>>24)&0xFF] ^ rk[4*r+0];
            t1 = d_T0[s1 & 0xFF] ^ d_T1[(s2>>8)&0xFF] ^ d_T2[(s3>>16)&0xFF] ^ d_T3[(s0>>24)&0xFF] ^ rk[4*r+1];
            t2 = d_T0[s2 & 0xFF] ^ d_T1[(s3>>8)&0xFF] ^ d_T2[(s0>>16)&0xFF] ^ d_T3[(s1>>24)&0xFF] ^ rk[4*r+2];
            t3 = d_T0[s3 & 0xFF] ^ d_T1[(s0>>8)&0xFF] ^ d_T2[(s1>>16)&0xFF] ^ d_T3[(s2>>24)&0xFF] ^ rk[4*r+3];
            s0=t0; s1=t1; s2=t2; s3=t3;
        }
        uint8_t keystream[16];
        uint8_t *ko = keystream;
        const uint8_t *sb = d_sbox;
        ko[0]  = sb[s0 & 0xFF];       ko[4]  = sb[(s1>>8)&0xFF];
        ko[8]  = sb[(s2>>16)&0xFF];   ko[12] = sb[(s3>>24)&0xFF];
        ko[1]  = sb[s1 & 0xFF];       ko[5]  = sb[(s2>>8)&0xFF];
        ko[9]  = sb[(s3>>16)&0xFF];   ko[13] = sb[(s0>>24)&0xFF];
        ko[2]  = sb[s2 & 0xFF];       ko[6]  = sb[(s3>>8)&0xFF];
        ko[10] = sb[(s0>>16)&0xFF];   ko[14] = sb[(s1>>24)&0xFF];
        ko[3]  = sb[s3 & 0xFF];       ko[7]  = sb[(s0>>8)&0xFF];
        ko[11] = sb[(s1>>16)&0xFF];   ko[15] = sb[(s2>>24)&0xFF];
        ((uint32_t*)keystream)[0] ^= rk[56];
        ((uint32_t*)keystream)[1] ^= rk[57];
        ((uint32_t*)keystream)[2] ^= rk[58];
        ((uint32_t*)keystream)[3] ^= rk[59];
        const uint8_t *ct_block = cipher + i * 16;
        uint8_t *pt_block = plain + i * 16;
        ((uint32_t*)pt_block)[0] = ((const uint32_t*)ct_block)[0] ^ ((uint32_t*)keystream)[0];
        ((uint32_t*)pt_block)[1] = ((const uint32_t*)ct_block)[1] ^ ((uint32_t*)keystream)[1];
        ((uint32_t*)pt_block)[2] = ((const uint32_t*)ct_block)[2] ^ ((uint32_t*)keystream)[2];
        ((uint32_t*)pt_block)[3] = ((const uint32_t*)ct_block)[3] ^ ((uint32_t*)keystream)[3];
    }
    __syncthreads();

    uint32_t tid = threadIdx.x;
    uint64_t partial_h = 0ull, partial_l = 0ull;
    for (size_t i = tid; i < nBlocks; i += blockDim.x) {
        uint64_t c_l = ((uint64_t*)cipher)[2*i + 0];
        uint64_t c_h = ((uint64_t*)cipher)[2*i + 1];
        size_t exp = nBlocks - i;
        uint64_t pow_h = 0ull, pow_l = 1ull;
        for (int b=0; b<MAX_POW_BITS; ++b) {
            if (exp & (1u<<b)) {
                gf_mul128(pow_h, pow_l, pow_H_hi[b], pow_H_lo[b]);
            }
        }
        gf_mul128(c_h, c_l, pow_h, pow_l);
        partial_h ^= c_h;
        partial_l ^= c_l;
    }
    partial_tag_hi[tid] = partial_h;
    partial_tag_lo[tid] = partial_l;
    __syncthreads();
    for (unsigned s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            partial_tag_hi[tid] ^= partial_tag_hi[tid + s];
            partial_tag_lo[tid] ^= partial_tag_lo[tid + s];
        }
        __syncthreads();
    }
    if (tid == 0) {
        ((uint64_t*)tagOut)[0] = partial_tag_lo[0];
        ((uint64_t*)tagOut)[1] = partial_tag_hi[0];
    }
}
