#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <random>
#include <fstream>
#include <filesystem>
#include <chrono>
#include <cstring>
#include <getopt.h>
#include <immintrin.h>
#include <openssl/evp.h>
#include "aes_common.h"
#include "profiling_helpers.h"

// -------------------------------
// Error handling macro
// -------------------------------
#define CHECK_CUDA(x) do { \
    hipError_t err = (x); \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        std::exit(EXIT_FAILURE); \
    } \
} while (0)

// -------------------------------
// Constants and parameters
// -------------------------------
constexpr int THREADS_PER_BLOCK = 256;
constexpr int NUM_RUNS          = 5;
static const size_t SIZES[]     = {1ull<<20, 10ull<<20, 100ull<<20, 1ull<<30};
static const char*  MODES[]     = {"ecb-128","ecb-256","ctr-128","ctr-256","gcm-128","gcm-256"};

// -------------------------------
// CTR helper
// -------------------------------
static void packCtr(const uint8_t iv[12], uint64_t &lo, uint64_t &hi) {
    uint32_t w0=0,w1=0,w2=0; memcpy(&w0,iv,4); memcpy(&w1,iv+4,4); memcpy(&w2,iv+8,4);
    uint32_t w3=0x01000000u; lo = (uint64_t)w0 | ((uint64_t)w1<<32); hi = (uint64_t)w2 | ((uint64_t)w3<<32);
}

// -------------------------------
// Device GF multiply used for --gf-mult and GCM debug
// -------------------------------
__device__ inline void gf_mul128_dev(uint64_t &Ah, uint64_t &Al, uint64_t Bh, uint64_t Bl) {
    uint64_t Zh=0, Zl=0, Vh=Bh, Vl=Bl; const uint64_t R=0xE100000000000000ULL;
    for(int i=0;i<128;++i){
        if(Al & 1ULL){ Zl^=Vl; Zh^=Vh; }
        bool carry = Vl & 1ULL;
        Vl = (Vl>>1) | (Vh<<63); Vh >>=1; if(carry) Vh^=R;
        Al = (Al>>1) | (Ah<<63); Ah >>=1;
    }
    Ah=Zh; Al=Zl;
}

// Kernel performing many GF multiplies per thread
__global__ void gf_mult_kernel(uint64_t *out) {
    uint64_t Ah=0x0123456789abcdefULL, Al=0xfedcba9876543210ULL;
    uint64_t Bh=0x0fedcba987654321ULL, Bl=0x1234567890abcdefULL;
    for(int i=0;i<1000000;i++) {
        gf_mul128_dev(Ah,Al,Bh,Bl);
        Bh += 1; Bl += 1;
    }
    out[threadIdx.x] = Ah ^ Al ^ Bh ^ Bl;
}

// Kernel computing per-thread partial GHASH
__global__ void gcm_partial_kernel(const uint8_t *cipher, size_t nBlocks,
                                   uint64_t Hh, uint64_t Hl,
                                   uint64_t *outH, uint64_t *outL) {
    int tid = threadIdx.x;
    size_t start = tid * nBlocks / blockDim.x;
    size_t end   = (tid+1) * nBlocks / blockDim.x;
    uint64_t Xh=0, Xl=0;
    for(size_t i=start;i<end;++i){
        uint64_t cl=((const uint64_t*)cipher)[2*i];
        uint64_t ch=((const uint64_t*)cipher)[2*i+1];
        Xl ^= cl; Xh ^= ch; gf_mul128_dev(Xh,Xl,Hh,Hl);
    }
    outH[tid]=Xh; outL[tid]=Xl;
}

// -------------------------------
// OpenSSL throughput helper
// -------------------------------
static double cpu_aes_throughput(const void* src, size_t bytes,
                                 const unsigned char* key, int bits,
                                 bool decrypt, const EVP_CIPHER* (*cipherSel)()) {
    std::vector<unsigned char> buf(bytes);
    std::vector<unsigned char> iv(16,0);
    EVP_CIPHER_CTX *ctx = EVP_CIPHER_CTX_new();
    const EVP_CIPHER *cipher = cipherSel();
    if(decrypt) EVP_DecryptInit_ex(ctx, cipher, nullptr, key, iv.data());
    else        EVP_EncryptInit_ex(ctx, cipher, nullptr, key, iv.data());
    EVP_CIPHER_CTX_set_padding(ctx,0);
    auto t0=std::chrono::high_resolution_clock::now();
    int outLen=0,total=0;
    if(decrypt) EVP_DecryptUpdate(ctx, buf.data(), &outLen, (const unsigned char*)src, (int)bytes);
    else        EVP_EncryptUpdate(ctx, buf.data(), &outLen, (const unsigned char*)src, (int)bytes);
    total += outLen;
    if(decrypt) EVP_DecryptFinal_ex(ctx, buf.data()+total, &outLen);
    else        EVP_EncryptFinal_ex(ctx, buf.data()+total, &outLen);
    total += outLen;
    auto t1=std::chrono::high_resolution_clock::now();
    EVP_CIPHER_CTX_free(ctx);
    double ms=std::chrono::duration<double,std::milli>(t1-t0).count();
    double gib=(double)bytes/(double)(1ull<<30);
    return gib/(ms/1000.0);
}

// -------------------------------
// Helper to generate random bytes
// -------------------------------
static void fill_random(uint8_t *buf, size_t n, std::mt19937_64 &rng) {
    for(size_t i=0;i<n;++i) buf[i] = static_cast<uint8_t>(rng() & 0xFF);
}

// -------------------------------
// CTR preview routine
// -------------------------------
static int ctr_preview() {
    std::mt19937_64 rng(42);
    std::vector<uint8_t> key(16); fill_random(key.data(),16,rng);
    std::vector<uint8_t> iv(12);  fill_random(iv.data(),12,rng);

    std::vector<uint32_t> rk(44); expandKey128(key.data(), rk.data());
    init_roundKeys(rk.data(), (int)rk.size());

    uint8_t *d_in,*d_out; CHECK_CUDA(hipMalloc(&d_in,32)); CHECK_CUDA(hipMalloc(&d_out,32));
    CHECK_CUDA(hipMemset(d_in,0,32));
    uint64_t lo=0,hi=0; packCtr(iv.data(),lo,hi);
    NVTX_PUSH("CTR_PREVIEW");
    aes128_ctr_encrypt<<<1,1>>>(d_in,d_out,2,lo,hi);
    CHECK_CUDA(hipDeviceSynchronize());
    NVTX_POP();
    uint8_t h_out[32]; CHECK_CUDA(hipMemcpy(h_out,d_out,32,hipMemcpyDeviceToHost));
    CHECK_CUDA(hipFree(d_in)); CHECK_CUDA(hipFree(d_out));

    printf("CTR_PREVIEW,");
    for(int i=0;i<32;i++){ printf("%02x", h_out[i]); if(i==15) printf(","); }
    printf("\n");
    return 0;
}

// -------------------------------
// GF multiply benchmark
// -------------------------------
static int gf_mult_bench() {
    std::filesystem::create_directories("bench");
    // CPU part
    double ms_cpu=0.0; {
        __m128i a = _mm_set_epi64x(0x0123456789abcdefULL,0xfedcba9876543210ULL);
        __m128i b = _mm_set_epi64x(0x0fedcba987654321ULL,0x1234567890abcdefULL);
        auto t0=std::chrono::high_resolution_clock::now();
        for(int i=0;i<1000000;i++) {
            __m128i r = _mm_clmulepi64_si128(a,b,0x00);
            a = _mm_xor_si128(a,r);
            b = _mm_xor_si128(b,r);
        }
        auto t1=std::chrono::high_resolution_clock::now();
        ms_cpu=std::chrono::duration<double,std::milli>(t1-t0).count();
    }
    double gbps_cpu = (1000000.0*128/1e9) / (ms_cpu/1000.0);

    // GPU part
    double ms_gpu=0.0; double gbps_gpu=0.0; {
        uint64_t *d_out; CHECK_CUDA(hipMalloc(&d_out, THREADS_PER_BLOCK*sizeof(uint64_t)));
        hipEvent_t s,e; hipEventCreate(&s); hipEventCreate(&e);
        hipEventRecord(s);
        gf_mult_kernel<<<1,THREADS_PER_BLOCK>>>(d_out);
        hipEventRecord(e); CHECK_CUDA(hipEventSynchronize(e));
        hipEventElapsedTime(&ms_gpu,s,e); CHECK_CUDA(hipFree(d_out));
        gbps_gpu = (1000000.0*THREADS_PER_BLOCK*128/1e9) / (ms_gpu/1000.0);
    }

    std::ofstream f("bench/gf_mult.csv", std::ios::app);
    f << "SRC,CPU,1000000," << ms_cpu << ',' << gbps_cpu << "\n";
    f << "SRC,GPU," << (1000000*THREADS_PER_BLOCK) << ',' << ms_gpu << ',' << gbps_gpu << "\n";
    std::cout << "GF_MULT CPU "<<gbps_cpu<<" Gbps\n";
    std::cout << "GF_MULT GPU "<<gbps_gpu<<" Gbps\n";
    return 0;
}

// -------------------------------
// GCM debug routine: encrypt 64B and dump partial GHASH
// -------------------------------
static int gcm_debug_run() {
    std::filesystem::create_directories("bench");
    std::mt19937_64 rng(123);
    const size_t bytes=64; size_t nBlocks=bytes/16;
    uint8_t *h_plain,*h_cipher; CHECK_CUDA(hipHostMalloc(&h_plain,bytes)); CHECK_CUDA(hipHostMalloc(&h_cipher,bytes));
    fill_random(h_plain,bytes,rng);
    std::vector<uint8_t> key(16); fill_random(key.data(),16,rng);
    std::vector<uint8_t> iv(12);  fill_random(iv.data(),12,rng);
    std::vector<uint32_t> rk(44); expandKey128(key.data(), rk.data());
    init_roundKeys(rk.data(), (int)rk.size());
    uint8_t *d_plain,*d_cipher,*d_tag,*d_iv; CHECK_CUDA(hipMalloc(&d_plain,bytes)); CHECK_CUDA(hipMalloc(&d_cipher,bytes)); CHECK_CUDA(hipMalloc(&d_tag,16)); CHECK_CUDA(hipMalloc(&d_iv,12));
    CHECK_CUDA(hipMemcpy(d_plain,h_plain,bytes,hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_iv,iv.data(),12,hipMemcpyHostToDevice));
    aes128_gcm_encrypt<<<1,THREADS_PER_BLOCK>>>(d_plain,d_cipher,nBlocks,d_iv,d_tag);
    CHECK_CUDA(hipDeviceSynchronize());
    CHECK_CUDA(hipMemcpy(h_cipher,d_cipher,bytes,hipMemcpyDeviceToHost));

    // compute H = AES_k(0)
    uint8_t *d_zero,*d_h; CHECK_CUDA(hipMalloc(&d_zero,16)); CHECK_CUDA(hipMalloc(&d_h,16));
    CHECK_CUDA(hipMemset(d_zero,0,16));
    aes128_ecb_encrypt<<<1,1>>>(d_zero,d_h,1);
    CHECK_CUDA(hipDeviceSynchronize());
    uint8_t hbuf[16]; CHECK_CUDA(hipMemcpy(hbuf,d_h,16,hipMemcpyDeviceToHost));
    uint64_t Hl=((uint64_t*)hbuf)[0]; uint64_t Hh=((uint64_t*)hbuf)[1];
    CHECK_CUDA(hipFree(d_zero)); CHECK_CUDA(hipFree(d_h));

    // partial GHASH
    uint64_t *d_ph,*d_pl; CHECK_CUDA(hipMalloc(&d_ph,THREADS_PER_BLOCK*sizeof(uint64_t))); CHECK_CUDA(hipMalloc(&d_pl,THREADS_PER_BLOCK*sizeof(uint64_t)));
    gcm_partial_kernel<<<1,THREADS_PER_BLOCK>>>(d_cipher,nBlocks,Hh,Hl,d_ph,d_pl);
    CHECK_CUDA(hipDeviceSynchronize());
    std::vector<uint64_t> ph(THREADS_PER_BLOCK), pl(THREADS_PER_BLOCK);
    CHECK_CUDA(hipMemcpy(ph.data(),d_ph,THREADS_PER_BLOCK*sizeof(uint64_t),hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(pl.data(),d_pl,THREADS_PER_BLOCK*sizeof(uint64_t),hipMemcpyDeviceToHost));
    CHECK_CUDA(hipFree(d_ph)); CHECK_CUDA(hipFree(d_pl));
    std::ofstream out("bench/ghash_partials.txt");
    for(int i=0;i<THREADS_PER_BLOCK;i++)
        out << i << "," << std::hex << ph[i] << "," << pl[i] << std::dec << "\n";

    CHECK_CUDA(hipFree(d_plain)); CHECK_CUDA(hipFree(d_cipher)); CHECK_CUDA(hipFree(d_tag)); CHECK_CUDA(hipFree(d_iv));
    CHECK_CUDA(hipHostFree(h_plain)); CHECK_CUDA(hipHostFree(h_cipher));
    std::cout << "GHASH partials written to bench/ghash_partials.txt\n";
    return 0;
}

// -------------------------------
// Main benchmark loop
// -------------------------------
int main(int argc, char** argv) {
    int blockOverride = THREADS_PER_BLOCK;
    bool decrypt=false, doCtrPreview=false, doGcmDebug=false, doGfMult=false;

    enum { OPT_CTR_PREVIEW=1000, OPT_GCM_DEBUG, OPT_GF_MULT };
    static struct option opts[] = {
        {"block", required_argument, nullptr, 'b'},
        {"decrypt", no_argument, nullptr, 'd'},
        {"ctr-preview", no_argument, nullptr, OPT_CTR_PREVIEW},
        {"gcm-debug", no_argument, nullptr, OPT_GCM_DEBUG},
        {"gf-mult", no_argument, nullptr, OPT_GF_MULT},
        {"help", no_argument, nullptr, 'h'},
        {0,0,0,0}
    };
    while(true){
        int idx=0; int c=getopt_long(argc,argv,"b:dh",opts,&idx); if(c==-1) break;
        switch(c){
            case 'b': blockOverride=atoi(optarg); break;
            case 'd': decrypt=true; break;
            case OPT_CTR_PREVIEW: doCtrPreview=true; break;
            case OPT_GCM_DEBUG: doGcmDebug=true; break;
            case OPT_GF_MULT: doGfMult=true; break;
            case 'h':
            default:
                std::cout << "Usage: "<<argv[0]<<" [--block N] [--decrypt] [--ctr-preview] [--gcm-debug] [--gf-mult]\n";
                return 0;
        }
    }

    std::filesystem::create_directories("bench");
    init_T_tables();

    if(doCtrPreview) return ctr_preview();
    if(doGcmDebug)   return gcm_debug_run();
    if(doGfMult)     return gf_mult_bench();

    std::mt19937_64 rng(12345);
    for(const char* modeStr : MODES){
        std::string mode(modeStr);
        bool isEcb = mode.find("ecb")==0;
        bool isCtr = mode.find("ctr")==0;
        bool isGcm = mode.find("gcm")==0;
        int bits = mode.find("256")!=std::string::npos ? 256 : 128;
        size_t keyBytes = bits/8;
        std::vector<uint8_t> key(keyBytes); fill_random(key.data(),keyBytes,rng);
        std::vector<uint32_t> rk(bits==128?44:60);
        if(bits==128) expandKey128(key.data(),rk.data()); else expandKey256(key.data(),rk.data());
        init_roundKeys(rk.data(), (int)rk.size());
        std::vector<uint8_t> iv(12); if(!isEcb) fill_random(iv.data(),12,rng);

        for(size_t sz : SIZES){
            size_t nBlocks=(sz+15)/16; size_t bytes=nBlocks*16;
            for(int run=1; run<=NUM_RUNS; ++run){
                uint8_t *h_in,*h_out; CHECK_CUDA(hipHostMalloc(&h_in,bytes)); CHECK_CUDA(hipHostMalloc(&h_out,bytes));
                fill_random(h_in,bytes,rng);
                uint8_t *d_in,*d_out,*d_tag=nullptr,*d_iv=nullptr;
                CHECK_CUDA(hipMalloc(&d_in,bytes)); CHECK_CUDA(hipMalloc(&d_out,bytes));
                if(isGcm) { CHECK_CUDA(hipMalloc(&d_tag,16)); CHECK_CUDA(hipMalloc(&d_iv,12)); CHECK_CUDA(hipMemcpy(d_iv,iv.data(),12,hipMemcpyHostToDevice)); }
                CHECK_CUDA(hipMemcpy(d_in,h_in,bytes,hipMemcpyHostToDevice));
                dim3 block(blockOverride); dim3 grid((unsigned)((nBlocks+block.x-1)/block.x));
                hipEvent_t s,e; hipEventCreate(&s); hipEventCreate(&e);
                hipEventRecord(s);
                if(!decrypt){
                    if(isEcb && bits==128){ NVTX_PUSH("ecb128_enc"); aes128_ecb_encrypt<<<grid,block>>>(d_in,d_out,nBlocks); NVTX_POP(); }
                    else if(isEcb && bits==256){ NVTX_PUSH("ecb256_enc"); aes256_ecb_encrypt<<<grid,block>>>(d_in,d_out,nBlocks); NVTX_POP(); }
                    else if(isCtr && bits==128){ uint64_t lo,hi; packCtr(iv.data(),lo,hi); NVTX_PUSH("ctr128_enc"); aes128_ctr_encrypt<<<grid,block>>>(d_in,d_out,nBlocks,lo,hi); NVTX_POP(); }
                    else if(isCtr && bits==256){ uint64_t lo,hi; packCtr(iv.data(),lo,hi); NVTX_PUSH("ctr256_enc"); aes256_ctr_encrypt<<<grid,block>>>(d_in,d_out,nBlocks,lo,hi); NVTX_POP(); }
                    else if(isGcm && bits==128){ NVTX_PUSH("gcm128_enc"); aes128_gcm_encrypt<<<1,THREADS_PER_BLOCK>>>(d_in,d_out,nBlocks,d_iv,d_tag); NVTX_POP(); }
                    else if(isGcm && bits==256){ NVTX_PUSH("gcm256_enc"); aes256_gcm_encrypt<<<1,THREADS_PER_BLOCK>>>(d_in,d_out,nBlocks,d_iv,d_tag); NVTX_POP(); }
                } else {
                    if(isEcb && bits==128){ NVTX_PUSH("ecb128_dec"); aes128_ecb_decrypt<<<grid,block>>>(d_in,d_out,nBlocks); NVTX_POP(); }
                    else if(isEcb && bits==256){ NVTX_PUSH("ecb256_dec"); aes256_ecb_decrypt<<<grid,block>>>(d_in,d_out,nBlocks); NVTX_POP(); }
                    else if(isCtr && bits==128){ uint64_t lo,hi; packCtr(iv.data(),lo,hi); NVTX_PUSH("ctr128_dec"); aes128_ctr_decrypt<<<grid,block>>>(d_in,d_out,nBlocks,lo,hi); NVTX_POP(); }
                    else if(isCtr && bits==256){ uint64_t lo,hi; packCtr(iv.data(),lo,hi); NVTX_PUSH("ctr256_dec"); aes256_ctr_decrypt<<<grid,block>>>(d_in,d_out,nBlocks,lo,hi); NVTX_POP(); }
                    else if(isGcm && bits==128){ NVTX_PUSH("gcm128_dec"); aes128_gcm_decrypt<<<1,THREADS_PER_BLOCK>>>(d_in,d_out,nBlocks,d_iv,d_tag,d_tag); NVTX_POP(); }
                    else if(isGcm && bits==256){ NVTX_PUSH("gcm256_dec"); aes256_gcm_decrypt<<<1,THREADS_PER_BLOCK>>>(d_in,d_out,nBlocks,d_iv,d_tag,d_tag); NVTX_POP(); }
                }
                hipEventRecord(e); CHECK_CUDA(hipEventSynchronize(e)); float ms=0.0f; hipEventElapsedTime(&ms,s,e);
                double gib=(double)bytes/(double)(1ull<<30); double thr=gib/(ms/1000.0);
                printf("RESULT_GPU,%s,%zu,%d,%.3f,%.3f,%s\n", mode.c_str(), bytes, run, ms, thr, decrypt?"DEC":"ENC");
                std::ofstream fg("bench/thr_gpu.csv",std::ios::app); fg<<"RESULT_GPU,"<<mode<<","<<bytes<<","<<run<<","<<ms<<","<<thr<<","<<(decrypt?"DEC":"ENC")<<"\n";

                std::vector<uint8_t> host_in(bytes); CHECK_CUDA(hipMemcpy(host_in.data(),d_in,bytes,hipMemcpyDeviceToHost));
                const EVP_CIPHER* (*sel)();
                if(isEcb&&bits==128) sel=&EVP_aes_128_ecb; else if(isEcb&&bits==256) sel=&EVP_aes_256_ecb;
                else if(isCtr&&bits==128) sel=&EVP_aes_128_ctr; else if(isCtr&&bits==256) sel=&EVP_aes_256_ctr;
                else if(isGcm&&bits==128) sel=&EVP_aes_128_gcm; else sel=&EVP_aes_256_gcm;
                double cpu_thr = cpu_aes_throughput(host_in.data(), bytes, key.data(), bits, decrypt, sel);
                double ms_cpu = (double)bytes/(cpu_thr*(1ull<<30))*1000.0;
                printf("RESULT_CPU,%s,%zu,%d,%.3f,%.3f,%s\n", mode.c_str(), bytes, run, ms_cpu, cpu_thr, decrypt?"DEC":"ENC");
                std::ofstream fc("bench/thr_cpu.csv",std::ios::app); fc<<"RESULT_CPU,"<<mode<<","<<bytes<<","<<run<<","<<ms_cpu<<","<<cpu_thr<<","<<(decrypt?"DEC":"ENC")<<"\n";

                CHECK_CUDA(hipHostFree(h_in)); CHECK_CUDA(hipHostFree(h_out));
                CHECK_CUDA(hipFree(d_in)); CHECK_CUDA(hipFree(d_out)); if(d_tag) CHECK_CUDA(hipFree(d_tag)); if(d_iv) CHECK_CUDA(hipFree(d_iv));
            }
        }
    }
    return 0;
}

/*
BUILD:
cmake -S . -B build -G "Ninja" -DCMAKE_BUILD_TYPE=Release
cmake --build build --config Release -j

BASELINE RUN (all figures R-1..R-3):
build/CudaProject.exe

OCCUPANCY SWEEP (M-2, D-3):
for %B in (32 64 128 256 512) do build\CudaProject.exe --block %B

CTR COUNTER (I-4):
build/CudaProject.exe --ctr-preview

GHASH PARTIALS (M-3):
build/CudaProject.exe --gcm-debug

GF MULTIPLY (D-2):
build/CudaProject.exe --gf-mult

NSYS TIMELINE + MEM-TRACE (M-1 & M-4):
nsys profile --trace cuda,nvtx -o bench/run build/CudaProject.exe

ROOFLINE METRICS (M-5, D-1):
ncu --metrics flop_count_sp,dram__bytes_read.sum,dram__bytes_write.sum
--set full --csv --target-processes all build/CudaProject.exe

PTX DUMPS (D-5):
nvdisasm --print-code aes128_ecb.cu.obj > bench/ptx_lookup.txt
*/
