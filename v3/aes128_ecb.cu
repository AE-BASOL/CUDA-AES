/********************************************************************
 *  aes128_ecb.cu  – one-thread-per-block AES-128 ECB encryptor
 *                   **little-endian state + tables**
 *******************************************************************/
#include <hip/hip_runtime.h>
#include <cstdint>

extern __device__ __constant__ uint32_t d_roundKeys[60];
extern __device__ __constant__ uint32_t d_T0[256], d_T1[256],
                                       d_T2[256], d_T3[256];
extern __device__ __constant__ uint8_t  d_sbox[256];
extern __device__ __constant__ uint8_t  d_inv_sbox[256];
extern __device__ __constant__ uint32_t d_U0[256], d_U1[256], d_U2[256], d_U3[256];


/* ------------ single round (little-endian tables) ------------- */
#define AES_ROUND(o0,o1,o2,o3,s0,s1,s2,s3,rk)                    \
{                                                                \
    (o0)=sh_T0[(s0      ) &0xFF] ^ sh_T1[((s1>>  8)&0xFF)] ^     \
         sh_T2[((s2>>16)&0xFF)] ^ sh_T3[((s3>>24)&0xFF)] ^ (rk)[0];\
    (o1)=sh_T0[(s1      ) &0xFF] ^ sh_T1[((s2>>  8)&0xFF)] ^     \
         sh_T2[((s3>>16)&0xFF)] ^ sh_T3[((s0>>24)&0xFF)] ^ (rk)[1];\
    (o2)=sh_T0[(s2      ) &0xFF] ^ sh_T1[((s3>>  8)&0xFF)] ^     \
         sh_T2[((s0>>16)&0xFF)] ^ sh_T3[((s1>>24)&0xFF)] ^ (rk)[2];\
    (o3)=sh_T0[(s3      ) &0xFF] ^ sh_T1[((s0>>  8)&0xFF)] ^     \
         sh_T2[((s1>>16)&0xFF)] ^ sh_T3[((s2>>24)&0xFF)] ^ (rk)[3];\
}

/* ====================== kernel ============================== */
__global__ void aes128_ecb_encrypt(const uint8_t *in,
                                   uint8_t       *out,
                                   size_t         nBlocks)
{
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= nBlocks) return;

    __shared__ uint32_t sh_T0[256], sh_T1[256], sh_T2[256], sh_T3[256];
    __shared__ uint8_t  sh_sbox[256];
    for (int i = threadIdx.x; i < 256; i += blockDim.x) {
        sh_T0[i] = d_T0[i];
        sh_T1[i] = d_T1[i];
        sh_T2[i] = d_T2[i];
        sh_T3[i] = d_T3[i];
        sh_sbox[i] = d_sbox[i];
    }
    __syncthreads();

    const uint32_t *rk = d_roundKeys;         // 44 words

    /* ---- load 16 input bytes as 4 *little-endian* words ---- */
    uint4 block = reinterpret_cast<const uint4*>(in)[idx];
    uint32_t s0 = block.x;
    uint32_t s1 = block.y;
    uint32_t s2 = block.z;
    uint32_t s3 = block.w;

    /* initial AddRoundKey */
    s0 ^= rk[0];  s1 ^= rk[1];
    s2 ^= rk[2];  s3 ^= rk[3];

    /* 9 full rounds */
    uint32_t t0,t1,t2,t3;
#pragma unroll
    for (int r = 1; r <= 9; ++r) {
        AES_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk + 4*r);
        s0=t0; s1=t1; s2=t2; s3=t3;
    }

    /* final round (SubBytes + ShiftRows) */
    const uint8_t *sb = sh_sbox;
    uint32_t r0 = ((uint32_t)sb[ s0        & 0xFF]) |
                  ((uint32_t)sb[ s1        & 0xFF] << 8) |
                  ((uint32_t)sb[ s2        & 0xFF] << 16) |
                  ((uint32_t)sb[ s3        & 0xFF] << 24);
    uint32_t r1 = ((uint32_t)sb[(s1 >>  8) & 0xFF]) |
                  ((uint32_t)sb[(s2 >>  8) & 0xFF] << 8) |
                  ((uint32_t)sb[(s3 >>  8) & 0xFF] << 16) |
                  ((uint32_t)sb[(s0 >>  8) & 0xFF] << 24);
    uint32_t r2 = ((uint32_t)sb[(s2 >> 16) & 0xFF]) |
                  ((uint32_t)sb[(s3 >> 16) & 0xFF] << 8) |
                  ((uint32_t)sb[(s0 >> 16) & 0xFF] << 16) |
                  ((uint32_t)sb[(s1 >> 16) & 0xFF] << 24);
    uint32_t r3 = ((uint32_t)sb[(s3 >> 24) & 0xFF]) |
                  ((uint32_t)sb[(s0 >> 24) & 0xFF] << 8) |
                  ((uint32_t)sb[(s1 >> 24) & 0xFF] << 16) |
                  ((uint32_t)sb[(s2 >> 24) & 0xFF] << 24);

    /* final AddRoundKey */
    r0 ^= rk[40];
    r1 ^= rk[41];
    r2 ^= rk[42];
    r3 ^= rk[43];

    uint4 outBlock = make_uint4(r0, r1, r2, r3);
    reinterpret_cast<uint4*>(out)[idx] = outBlock;
}

__global__ void aes128_ecb_decrypt(const uint8_t *in,
                                   uint8_t       *out,
                                   size_t         nBlocks)
{
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= nBlocks) return;

    const uint32_t *rk = d_roundKeys;         // 44 words for AES-128
    const uint32_t *rk_last = rk + 40;        // last round key for AES-128

    // Load ciphertext and initial AddRoundKey with last round key
    uint4 inBlock = reinterpret_cast<const uint4*>(in)[idx];
    uint32_t s0 = inBlock.x ^ rk_last[0];
    uint32_t s1 = inBlock.y ^ rk_last[1];
    uint32_t s2 = inBlock.z ^ rk_last[2];
    uint32_t s3 = inBlock.w ^ rk_last[3];

    // 9 full inverse rounds (use U-tables, as AES-256)
    uint32_t t0, t1, t2, t3;
#pragma unroll
    for (int r = 9; r >= 1; --r) {
        t0 = d_U0[(s0      ) & 0xFF] ^ d_U1[(s3 >>  8) & 0xFF] ^
             d_U2[(s2 >> 16) & 0xFF] ^ d_U3[(s1 >> 24) & 0xFF] ^ rk[4*r + 0];
        t1 = d_U0[(s1      ) & 0xFF] ^ d_U1[(s0 >>  8) & 0xFF] ^
             d_U2[(s3 >> 16) & 0xFF] ^ d_U3[(s2 >> 24) & 0xFF] ^ rk[4*r + 1];
        t2 = d_U0[(s2      ) & 0xFF] ^ d_U1[(s1 >>  8) & 0xFF] ^
             d_U2[(s0 >> 16) & 0xFF] ^ d_U3[(s3 >> 24) & 0xFF] ^ rk[4*r + 2];
        t3 = d_U0[(s3      ) & 0xFF] ^ d_U1[(s2 >>  8) & 0xFF] ^
             d_U2[(s1 >> 16) & 0xFF] ^ d_U3[(s0 >> 24) & 0xFF] ^ rk[4*r + 3];
        s0 = t0; s1 = t1; s2 = t2; s3 = t3;
    }

    // Final round: InvShiftRows + InvSubBytes, then AddRoundKey
    const uint8_t *isbox = d_inv_sbox;
    uint32_t r0 = ((uint32_t)isbox[ s0        & 0xFF]) |
                  ((uint32_t)isbox[ s1        & 0xFF] << 8) |
                  ((uint32_t)isbox[ s2        & 0xFF] << 16) |
                  ((uint32_t)isbox[ s3        & 0xFF] << 24);
    uint32_t r1 = ((uint32_t)isbox[(s3 >>  8) & 0xFF]) |
                  ((uint32_t)isbox[(s0 >>  8) & 0xFF] << 8) |
                  ((uint32_t)isbox[(s1 >>  8) & 0xFF] << 16) |
                  ((uint32_t)isbox[(s2 >>  8) & 0xFF] << 24);
    uint32_t r2 = ((uint32_t)isbox[(s2 >> 16) & 0xFF]) |
                  ((uint32_t)isbox[(s3 >> 16) & 0xFF] << 8) |
                  ((uint32_t)isbox[(s0 >> 16) & 0xFF] << 16) |
                  ((uint32_t)isbox[(s1 >> 16) & 0xFF] << 24);
    uint32_t r3 = ((uint32_t)isbox[(s1 >> 24) & 0xFF]) |
                  ((uint32_t)isbox[(s2 >> 24) & 0xFF] << 8) |
                  ((uint32_t)isbox[(s3 >> 24) & 0xFF] << 16) |
                  ((uint32_t)isbox[(s0 >> 24) & 0xFF] << 24);

    r0 ^= rk[0];
    r1 ^= rk[1];
    r2 ^= rk[2];
    r3 ^= rk[3];

    uint4 outBlock = make_uint4(r0, r1, r2, r3);
    reinterpret_cast<uint4*>(out)[idx] = outBlock;
}

