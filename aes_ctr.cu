#include "hip/hip_runtime.h"
#include "aes_common.h"

extern __device__ __constant__ uint32_t d_roundKeys[60];
extern __device__ __constant__ uint32_t d_T0[256], d_T1[256], d_T2[256], d_T3[256];
extern __device__ __constant__ uint8_t  d_sbox[256];

#define CTR_ROUND(o0,o1,o2,o3,s0,s1,s2,s3,rk) do { \
    o0 = sh_T0[(s0)&0xFF] ^ sh_T1[((s1)>>8)&0xFF] ^ \
         sh_T2[((s2)>>16)&0xFF] ^ sh_T3[((s3)>>24)&0xFF] ^ (rk)[0]; \
    o1 = sh_T0[(s1)&0xFF] ^ sh_T1[((s2)>>8)&0xFF] ^ \
         sh_T2[((s3)>>16)&0xFF] ^ sh_T3[((s0)>>24)&0xFF] ^ (rk)[1]; \
    o2 = sh_T0[(s2)&0xFF] ^ sh_T1[((s3)>>8)&0xFF] ^ \
         sh_T2[((s0)>>16)&0xFF] ^ sh_T3[((s1)>>24)&0xFF] ^ (rk)[2]; \
    o3 = sh_T0[(s3)&0xFF] ^ sh_T1[((s0)>>8)&0xFF] ^ \
         sh_T2[((s1)>>16)&0xFF] ^ sh_T3[((s2)>>24)&0xFF] ^ (rk)[3]; \
} while(0)

template<int ROUNDS>
__device__ inline void ctr_keystream(uint64_t ctr_lo, uint64_t ctr_hi,
                                     uint32_t &k0, uint32_t &k1, uint32_t &k2, uint32_t &k3,
                                     const uint32_t *rk, const uint8_t *sb,
                                     uint32_t *sh_T0, uint32_t *sh_T1,
                                     uint32_t *sh_T2, uint32_t *sh_T3) {
    uint32_t s0 = (uint32_t)ctr_lo;
    uint32_t s1 = (uint32_t)(ctr_lo >> 32);
    uint32_t s2 = (uint32_t)ctr_hi;
    uint32_t s3 = (uint32_t)(ctr_hi >> 32);
    s0 ^= rk[0]; s1 ^= rk[1]; s2 ^= rk[2]; s3 ^= rk[3];
    uint32_t t0,t1,t2,t3;
#pragma unroll
    for(int r=1; r<ROUNDS; ++r) {
        CTR_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk + 4*r);
        s0=t0; s1=t1; s2=t2; s3=t3;
    }
    k0 = ((uint32_t)sb[ s0        & 0xFF]) |
         ((uint32_t)sb[ s1        & 0xFF] << 8) |
         ((uint32_t)sb[ s2        & 0xFF] << 16) |
         ((uint32_t)sb[ s3        & 0xFF] << 24);
    k1 = ((uint32_t)sb[(s1 >>  8) & 0xFF]) |
         ((uint32_t)sb[(s2 >>  8) & 0xFF] << 8) |
         ((uint32_t)sb[(s3 >>  8) & 0xFF] << 16) |
         ((uint32_t)sb[(s0 >>  8) & 0xFF] << 24);
    k2 = ((uint32_t)sb[(s2 >> 16) & 0xFF]) |
         ((uint32_t)sb[(s3 >> 16) & 0xFF] << 8) |
         ((uint32_t)sb[(s0 >> 16) & 0xFF] << 16) |
         ((uint32_t)sb[(s1 >> 16) & 0xFF] << 24);
    k3 = ((uint32_t)sb[(s3 >> 24) & 0xFF]) |
         ((uint32_t)sb[(s0 >> 24) & 0xFF] << 8) |
         ((uint32_t)sb[(s1 >> 24) & 0xFF] << 16) |
         ((uint32_t)sb[(s2 >> 24) & 0xFF] << 24);
    k0 ^= rk[4*ROUNDS + 0];
    k1 ^= rk[4*ROUNDS + 1];
    k2 ^= rk[4*ROUNDS + 2];
    k3 ^= rk[4*ROUNDS + 3];
}

template<int ROUNDS>
__device__ void aes_ctr_encrypt_impl(const uint8_t* __restrict__ in,
                                     uint8_t* __restrict__ out,
                                     size_t nBlocks, uint64_t ctrLo, uint64_t ctrHi) {
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t stride = blockDim.x * gridDim.x;
    if (idx >= nBlocks) return;

    __shared__ uint32_t sh_T0[256], sh_T1[256], sh_T2[256], sh_T3[256];
    __shared__ uint8_t  sh_sbox[256];
    if (threadIdx.x < 256) {
        sh_T0[threadIdx.x] = d_T0[threadIdx.x];
        sh_T1[threadIdx.x] = d_T1[threadIdx.x];
        sh_T2[threadIdx.x] = d_T2[threadIdx.x];
        sh_T3[threadIdx.x] = d_T3[threadIdx.x];
        sh_sbox[threadIdx.x] = d_sbox[threadIdx.x];
    }
    __syncthreads();
    const uint32_t *rk = d_roundKeys;
    const uint8_t *sb = sh_sbox;

    uint64_t ctr_lo = ctrLo + idx;
    uint64_t ctr_hi = ctrHi + (ctr_lo < ctrLo);

    uint32_t k0,k1,k2,k3;
    ctr_keystream<ROUNDS>(ctr_lo, ctr_hi, k0,k1,k2,k3, rk, sb, sh_T0,sh_T1,sh_T2,sh_T3);
    uint4 inBlock = reinterpret_cast<const uint4*>(in)[idx];
    uint4 outBlock = make_uint4(inBlock.x ^ k0,
                                inBlock.y ^ k1,
                                inBlock.z ^ k2,
                                inBlock.w ^ k3);
    reinterpret_cast<uint4*>(out)[idx] = outBlock;

    size_t idx2 = idx + stride;
    if (idx2 < nBlocks) {
        ctr_lo = ctrLo + idx2;
        ctr_hi = ctrHi + (ctr_lo < ctrLo);
        ctr_keystream<ROUNDS>(ctr_lo, ctr_hi, k0,k1,k2,k3, rk, sb, sh_T0,sh_T1,sh_T2,sh_T3);
        inBlock = reinterpret_cast<const uint4*>(in)[idx2];
        outBlock = make_uint4(inBlock.x ^ k0,
                              inBlock.y ^ k1,
                              inBlock.z ^ k2,
                              inBlock.w ^ k3);
        reinterpret_cast<uint4*>(out)[idx2] = outBlock;
    }
}

__global__ void aes_ctr_encrypt_10(const uint8_t* __restrict__ in,
                                   uint8_t* __restrict__ out,
                                   size_t nBlocks,
                                   uint64_t ctrLo,
                                   uint64_t ctrHi) {
    aes_ctr_encrypt_impl<10>(in, out, nBlocks, ctrLo, ctrHi);
}

__global__ void aes_ctr_encrypt_14(const uint8_t* __restrict__ in,
                                   uint8_t* __restrict__ out,
                                   size_t nBlocks,
                                   uint64_t ctrLo,
                                   uint64_t ctrHi) {
    aes_ctr_encrypt_impl<14>(in, out, nBlocks, ctrLo, ctrHi);
}

template<int ROUNDS>
__device__ void aes_ctr_decrypt_impl(const uint8_t* __restrict__ in,
                                     uint8_t* __restrict__ out,
                                     size_t nBlocks, uint64_t ctrLo, uint64_t ctrHi) {
    aes_ctr_encrypt_impl<ROUNDS>(in, out, nBlocks, ctrLo, ctrHi);
}

__global__ void aes_ctr_decrypt_10(const uint8_t* __restrict__ in,
                                   uint8_t* __restrict__ out,
                                   size_t nBlocks,
                                   uint64_t ctrLo,
                                   uint64_t ctrHi) {
    aes_ctr_decrypt_impl<10>(in, out, nBlocks, ctrLo, ctrHi);
}

__global__ void aes_ctr_decrypt_14(const uint8_t* __restrict__ in,
                                   uint8_t* __restrict__ out,
                                   size_t nBlocks,
                                   uint64_t ctrLo,
                                   uint64_t ctrHi) {
    aes_ctr_decrypt_impl<14>(in, out, nBlocks, ctrLo, ctrHi);
}

// Explicit instantiation of device implementations used by the wrapper kernels
template __device__ void aes_ctr_encrypt_impl<10>(const uint8_t*, uint8_t*, size_t, uint64_t, uint64_t);
template __device__ void aes_ctr_encrypt_impl<14>(const uint8_t*, uint8_t*, size_t, uint64_t, uint64_t);
template __device__ void aes_ctr_decrypt_impl<10>(const uint8_t*, uint8_t*, size_t, uint64_t, uint64_t);
template __device__ void aes_ctr_decrypt_impl<14>(const uint8_t*, uint8_t*, size_t, uint64_t, uint64_t);
