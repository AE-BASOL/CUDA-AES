#include "hip/hip_runtime.h"
#include "aes_common.h"
#include <stdint.h>

// Extern constants
extern __device__ __constant__ uint32_t d_roundKeys[60];
extern __device__ __constant__ uint32_t d_T0[256], d_T1[256], d_T2[256], d_T3[256];
extern __device__ __constant__ uint8_t  d_sbox[256];
extern __device__ __constant__ uint64_t d_H_pow_hi[32];
extern __device__ __constant__ uint64_t d_H_pow_lo[32];

// Multiply two 128-bit values (A * B) in GF(2^128) with the GCM reduction polynomial.
// Inputs/outputs are in 64-bit high/low parts.
static __device__ inline void gf_mul128(uint64_t &Ah, uint64_t &Al, uint64_t Bh, uint64_t Bl) {
    // Implements bitwise multiplication with reduction: O(128) steps
    uint64_t Zh = 0ull, Zl = 0ull;
    uint64_t Vh = Bh, Vl = Bl;
    // GF(2^128) irreducible polynomial: x^128 + x^7 + x^2 + x + 1 (0xE100000000000000 as 128-bit)
    const uint64_t R = 0xE100000000000000ULL;  // Represents polynomial (1 followed by 0xE1)
    for (int i = 0; i < 128; ++i) {
        if (Al & 1ULL) {  // if LSB of A is 1, add V to Z
            Zl ^= Vl;
            Zh ^= Vh;
        }
        // Carry for reduction = LSB of V
        bool carry = (Vl & 1ULL);
        // Shift V right by 1
        Vl = (Vl >> 1) | (Vh << 63);
        Vh = (Vh >> 1);
        if (carry) {
            Vh ^= R;  // XOR high part with reduction constant if carry out
        }
        // Shift A right by 1 (to process next bit)
        bool a_carry = (Al & 1ULL);
        Al = (Al >> 1) | (Ah << 63);
        Ah = (Ah >> 1);
        // (We don't actually use a_carry here because we already used A's LSB for if)
    }
    Ah = Zh;
    Al = Zl;
}

// AES-128-GCM encryption kernel
__global__ void aes128_gcm_encrypt(const uint8_t * __restrict__ plain, uint8_t * __restrict__ cipher, size_t nBlocks, const uint8_t * __restrict__ iv, uint8_t * __restrict__ tagOut) {
    // We will use one thread block to process the entire message for tag calculation correctness (no inter-block sync on device).
    // Use 256 threads: parallelize CTR encryption, then do GHASH in a single warp for tag.
    // nBlocks is number of 16-byte blocks of plaintext.
    // GHASH uses precomputed H powers in constant memory; no setup needed

    // CTR mode encryption of plaintext
    // Prepare IV counter (96-bit IV + 32-bit initial counter=1)
    uint64_t IV_lo = 0ull;
    uint64_t IV_hi = 0ull;
    if (threadIdx.x == 0) {
        uint32_t w0 = 0, w1 = 0, w2 = 0;
        memcpy(&w0, iv, 4);
        memcpy(&w1, iv + 4, 4);
        memcpy(&w2, iv + 8, 4);
        uint32_t w3 = 0x01000000u; // counter = 1 in big-endian
        IV_lo = (uint64_t)w0 | ((uint64_t)w1 << 32);
        IV_hi = (uint64_t)w2 | ((uint64_t)w3 << 32);
    }
    __syncthreads();
    // Broadcast IV values to all threads
    IV_lo = __shfl_sync(0xFFFFFFFF, IV_lo, 0);
    IV_hi = __shfl_sync(0xFFFFFFFF, IV_hi, 0);

    // Parallel encryption: each thread handles multiple blocks in strides
    for (size_t i = threadIdx.x; i < nBlocks; i += blockDim.x) {
        // Compute counter for block i
        uint64_t ctr_lo = IV_lo + i;
        uint64_t ctr_hi = IV_hi;
        if (ctr_lo < IV_lo) ctr_hi += 1ULL;
        // Encrypt counter (like in CTR encryption)
        uint32_t s0 = (uint32_t) ctr_lo;
        uint32_t s1 = (uint32_t)(ctr_lo >> 32);
        uint32_t s2 = (uint32_t) ctr_hi;
        uint32_t s3 = (uint32_t)(ctr_hi >> 32);
        const uint32_t *rk = d_roundKeys;
        s0 ^= rk[0]; s1 ^= rk[1]; s2 ^= rk[2]; s3 ^= rk[3];
        uint32_t t0,t1,t2,t3;
        #pragma unroll
        for (int r = 1; r <= 9; ++r) {
            t0 = d_T0[s0 & 0xFF] ^ d_T1[(s1>>8)&0xFF] ^ d_T2[(s2>>16)&0xFF] ^ d_T3[(s3>>24)&0xFF] ^ rk[4*r+0];
            t1 = d_T0[s1 & 0xFF] ^ d_T1[(s2>>8)&0xFF] ^ d_T2[(s3>>16)&0xFF] ^ d_T3[(s0>>24)&0xFF] ^ rk[4*r+1];
            t2 = d_T0[s2 & 0xFF] ^ d_T1[(s3>>8)&0xFF] ^ d_T2[(s0>>16)&0xFF] ^ d_T3[(s1>>24)&0xFF] ^ rk[4*r+2];
            t3 = d_T0[s3 & 0xFF] ^ d_T1[(s0>>8)&0xFF] ^ d_T2[(s1>>16)&0xFF] ^ d_T3[(s2>>24)&0xFF] ^ rk[4*r+3];
            s0=t0; s1=t1; s2=t2; s3=t3;
        }
        // Final round
        uint8_t keystream[16];
        uint8_t *ko = keystream;
        const uint8_t *sb = d_sbox;
        ko[0]  = sb[s0 & 0xFF];        ko[4]  = sb[(s1>>8)&0xFF];
        ko[8]  = sb[(s2>>16)&0xFF];    ko[12] = sb[(s3>>24)&0xFF];
        ko[1]  = sb[s1 & 0xFF];        ko[5]  = sb[(s2>>8)&0xFF];
        ko[9]  = sb[(s3>>16)&0xFF];    ko[13] = sb[(s0>>24)&0xFF];
        ko[2]  = sb[s2 & 0xFF];        ko[6]  = sb[(s3>>8)&0xFF];
        ko[10] = sb[(s0>>16)&0xFF];    ko[14] = sb[(s1>>24)&0xFF];
        ko[3]  = sb[s3 & 0xFF];        ko[7]  = sb[(s0>>8)&0xFF];
        ko[11] = sb[(s1>>16)&0xFF];    ko[15] = sb[(s2>>24)&0xFF];
        ((uint32_t*)keystream)[0] ^= rk[40];
        ((uint32_t*)keystream)[1] ^= rk[41];
        ((uint32_t*)keystream)[2] ^= rk[42];
        ((uint32_t*)keystream)[3] ^= rk[43];
        // XOR plaintext with keystream to get ciphertext
        const uint8_t *pt_block = plain + i * 16;
        uint8_t *ct_block = cipher + i * 16;
        ((uint32_t*)ct_block)[0] = ((const uint32_t*)pt_block)[0] ^ ((uint32_t*)keystream)[0];
        ((uint32_t*)ct_block)[1] = ((const uint32_t*)pt_block)[1] ^ ((uint32_t*)keystream)[1];
        ((uint32_t*)ct_block)[2] = ((const uint32_t*)pt_block)[2] ^ ((uint32_t*)keystream)[2];
        ((uint32_t*)ct_block)[3] = ((const uint32_t*)pt_block)[3] ^ ((uint32_t*)keystream)[3];
    }
    __syncthreads();

    // GHASH computation using warp-level XOR reduction
    uint32_t tid = threadIdx.x;
    if (tid < 32) {
        // Precompute H^32 for stepping between iterations
        uint64_t step_hi = 0ull, step_lo = 1ull;
        for (int b = 0; b < 32; ++b)
            if (32u & (1u << b))
                gf_mul128(step_hi, step_lo, d_H_pow_hi[b], d_H_pow_lo[b]);

        // Starting power for this thread corresponds to H^(nBlocks-1-tid)
        uint64_t pow_hi = 0ull, pow_lo = 1ull;
        uint32_t exp = (uint32_t)(nBlocks - 1 - tid);
        for (int b = 0; b < 32; ++b)
            if (exp & (1u << b))
                gf_mul128(pow_hi, pow_lo, d_H_pow_hi[b], d_H_pow_lo[b]);

        uint64_t accum_hi = 0ull, accum_lo = 0ull;
        for (size_t j = tid; j < nBlocks; j += 32) {
            uint64_t c_lo = ((const uint64_t*)cipher)[2*j + 0];
            uint64_t c_hi = ((const uint64_t*)cipher)[2*j + 1];
            uint64_t tmp_hi = c_hi, tmp_lo = c_lo;
            gf_mul128(tmp_hi, tmp_lo, pow_hi, pow_lo);
            accum_hi ^= tmp_hi;
            accum_lo ^= tmp_lo;
            gf_mul128(pow_hi, pow_lo, step_hi, step_lo);  // advance power by 32
        }

        for (int off = 16; off > 0; off >>= 1) {
            accum_hi ^= __shfl_xor_sync(0xFFFFFFFF, accum_hi, off);
            accum_lo ^= __shfl_xor_sync(0xFFFFFFFF, accum_lo, off);
        }
        if (tid == 0) {
            uint64_t lenBlock_lo = (uint64_t)nBlocks * 16ull * 8ull;
            uint64_t lenBlock_hi = 0ull; // no AAD
            accum_lo ^= lenBlock_lo;
            accum_hi ^= lenBlock_hi;
            gf_mul128(accum_hi, accum_lo, d_H_pow_hi[0], d_H_pow_lo[0]);
            ((uint64_t*)tagOut)[0] = accum_lo;
            ((uint64_t*)tagOut)[1] = accum_hi;
        }
    }
}

__global__ void aes128_gcm_decrypt(const uint8_t * __restrict__ cipher, uint8_t * __restrict__ plain, size_t nBlocks, const uint8_t * __restrict__ iv, const uint8_t * __restrict__ tag, uint8_t * __restrict__ tagOut) {
    // Implement decryption directly rather than launching a nested kernel.
    // The operations mirror aes128_gcm_encrypt with plaintext and ciphertext
    // swapped.  The 'tag' parameter is provided for host-side verification and
    // is unused here.

    // GHASH uses precomputed powers of H

    // H subkey handled on host

    uint64_t IV_lo = 0ull, IV_hi = 0ull;
    if (threadIdx.x == 0) {
        uint32_t w0 = 0, w1 = 0, w2 = 0;
        memcpy(&w0, iv, 4);
        memcpy(&w1, iv + 4, 4);
        memcpy(&w2, iv + 8, 4);
        uint32_t w3 = 0x01000000u;
        IV_lo = (uint64_t)w0 | ((uint64_t)w1 << 32);
        IV_hi = (uint64_t)w2 | ((uint64_t)w3 << 32);
    }
    __syncthreads();
    IV_lo = __shfl_sync(0xFFFFFFFF, IV_lo, 0);
    IV_hi = __shfl_sync(0xFFFFFFFF, IV_hi, 0);

    for (size_t i = threadIdx.x; i < nBlocks; i += blockDim.x) {
        uint64_t ctr_lo = IV_lo + i;
        uint64_t ctr_hi = IV_hi;
        if (ctr_lo < IV_lo) ctr_hi += 1ULL;
        uint32_t s0 = (uint32_t) ctr_lo;
        uint32_t s1 = (uint32_t)(ctr_lo >> 32);
        uint32_t s2 = (uint32_t) ctr_hi;
        uint32_t s3 = (uint32_t)(ctr_hi >> 32);
        const uint32_t *rk = d_roundKeys;
        s0 ^= rk[0]; s1 ^= rk[1]; s2 ^= rk[2]; s3 ^= rk[3];
        uint32_t t0,t1,t2,t3;
#pragma unroll
        for (int r = 1; r <= 9; ++r) {
            t0 = d_T0[s0 & 0xFF] ^ d_T1[(s1>>8)&0xFF] ^ d_T2[(s2>>16)&0xFF] ^ d_T3[(s3>>24)&0xFF] ^ rk[4*r+0];
            t1 = d_T0[s1 & 0xFF] ^ d_T1[(s2>>8)&0xFF] ^ d_T2[(s3>>16)&0xFF] ^ d_T3[(s0>>24)&0xFF] ^ rk[4*r+1];
            t2 = d_T0[s2 & 0xFF] ^ d_T1[(s3>>8)&0xFF] ^ d_T2[(s0>>16)&0xFF] ^ d_T3[(s1>>24)&0xFF] ^ rk[4*r+2];
            t3 = d_T0[s3 & 0xFF] ^ d_T1[(s0>>8)&0xFF] ^ d_T2[(s1>>16)&0xFF] ^ d_T3[(s2>>24)&0xFF] ^ rk[4*r+3];
            s0=t0; s1=t1; s2=t2; s3=t3;
        }
        uint8_t keystream[16];
        uint8_t *ko = keystream;
        const uint8_t *sb = d_sbox;
        ko[0]  = sb[s0 & 0xFF];        ko[4]  = sb[(s1>>8)&0xFF];
        ko[8]  = sb[(s2>>16)&0xFF];    ko[12] = sb[(s3>>24)&0xFF];
        ko[1]  = sb[s1 & 0xFF];        ko[5]  = sb[(s2>>8)&0xFF];
        ko[9]  = sb[(s3>>16)&0xFF];    ko[13] = sb[(s0>>24)&0xFF];
        ko[2]  = sb[s2 & 0xFF];        ko[6]  = sb[(s3>>8)&0xFF];
        ko[10] = sb[(s0>>16)&0xFF];    ko[14] = sb[(s1>>24)&0xFF];
        ko[3]  = sb[s3 & 0xFF];        ko[7]  = sb[(s0>>8)&0xFF];
        ko[11] = sb[(s1>>16)&0xFF];    ko[15] = sb[(s2>>24)&0xFF];
        ((uint32_t*)keystream)[0] ^= rk[40];
        ((uint32_t*)keystream)[1] ^= rk[41];
        ((uint32_t*)keystream)[2] ^= rk[42];
        ((uint32_t*)keystream)[3] ^= rk[43];
        const uint8_t *ct_block = cipher + i * 16;
        uint8_t *pt_block = plain + i * 16;
        ((uint32_t*)pt_block)[0] = ((const uint32_t*)ct_block)[0] ^ ((uint32_t*)keystream)[0];
        ((uint32_t*)pt_block)[1] = ((const uint32_t*)ct_block)[1] ^ ((uint32_t*)keystream)[1];
        ((uint32_t*)pt_block)[2] = ((const uint32_t*)ct_block)[2] ^ ((uint32_t*)keystream)[2];
        ((uint32_t*)pt_block)[3] = ((const uint32_t*)ct_block)[3] ^ ((uint32_t*)keystream)[3];
    }
    __syncthreads();

    uint32_t tid = threadIdx.x;
    if (tid < 32) {
        uint64_t step_hi = 0ull, step_lo = 1ull;
        for (int b = 0; b < 32; ++b)
            if (32u & (1u << b))
                gf_mul128(step_hi, step_lo, d_H_pow_hi[b], d_H_pow_lo[b]);

        uint64_t pow_hi = 0ull, pow_lo = 1ull;
        uint32_t exp = (uint32_t)(nBlocks - 1 - tid);
        for (int b = 0; b < 32; ++b)
            if (exp & (1u << b))
                gf_mul128(pow_hi, pow_lo, d_H_pow_hi[b], d_H_pow_lo[b]);

        uint64_t accum_hi = 0ull, accum_lo = 0ull;
        for (size_t j = tid; j < nBlocks; j += 32) {
            uint64_t c_lo = ((const uint64_t*)cipher)[2*j + 0];
            uint64_t c_hi = ((const uint64_t*)cipher)[2*j + 1];
            uint64_t tmp_hi = c_hi, tmp_lo = c_lo;
            gf_mul128(tmp_hi, tmp_lo, pow_hi, pow_lo);
            accum_hi ^= tmp_hi;
            accum_lo ^= tmp_lo;
            gf_mul128(pow_hi, pow_lo, step_hi, step_lo);
        }

        for (int off = 16; off > 0; off >>= 1) {
            accum_hi ^= __shfl_xor_sync(0xFFFFFFFF, accum_hi, off);
            accum_lo ^= __shfl_xor_sync(0xFFFFFFFF, accum_lo, off);
        }
        if (tid == 0) {
            uint64_t lenBlock_lo = (uint64_t)nBlocks * 16ull * 8ull;
            uint64_t lenBlock_hi = 0ull;
            accum_lo ^= lenBlock_lo;
            accum_hi ^= lenBlock_hi;
            gf_mul128(accum_hi, accum_lo, d_H_pow_hi[0], d_H_pow_lo[0]);
            ((uint64_t*)tagOut)[0] = accum_lo;
            ((uint64_t*)tagOut)[1] = accum_hi;
        }
    }
}
