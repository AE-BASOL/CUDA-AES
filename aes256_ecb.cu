#include "hip/hip_runtime.h"
// AES-256 ECB kernels following the T-table strategy used in the CTR
// implementation and Cihangir Tezcan's optimized AES examples.  The
// interface remains identical but the internal rounds now rely on
// shared-memory T-tables for better performance.

#include <hip/hip_runtime.h>
#include <stdint.h>
#include "aes_common.h"

// Device constant memory declarations come from aes_common.h

// Macros implementing AES rounds with T-tables in shared memory
#define ENC_ROUND(o0,o1,o2,o3,s0,s1,s2,s3,rk)                          \
    do {                                                               \
        o0 = sh_T0[(s0)&0xFF] ^ sh_T1[((s1)>>8)&0xFF] ^               \
             sh_T2[((s2)>>16)&0xFF] ^ sh_T3[((s3)>>24)&0xFF] ^ (rk)[0];\
        o1 = sh_T0[(s1)&0xFF] ^ sh_T1[((s2)>>8)&0xFF] ^               \
             sh_T2[((s3)>>16)&0xFF] ^ sh_T3[((s0)>>24)&0xFF] ^ (rk)[1];\
        o2 = sh_T0[(s2)&0xFF] ^ sh_T1[((s3)>>8)&0xFF] ^               \
             sh_T2[((s0)>>16)&0xFF] ^ sh_T3[((s1)>>24)&0xFF] ^ (rk)[2];\
        o3 = sh_T0[(s3)&0xFF] ^ sh_T1[((s0)>>8)&0xFF] ^               \
             sh_T2[((s1)>>16)&0xFF] ^ sh_T3[((s2)>>24)&0xFF] ^ (rk)[3];\
    } while (0)

#define DEC_ROUND(o0,o1,o2,o3,s0,s1,s2,s3,rk)                          \
    do {                                                               \
        o0 = sh_U0[(s0)&0xFF] ^ sh_U1[((s3)>>8)&0xFF] ^               \
             sh_U2[((s2)>>16)&0xFF] ^ sh_U3[((s1)>>24)&0xFF] ^ (rk)[0];\
        o1 = sh_U0[(s1)&0xFF] ^ sh_U1[((s0)>>8)&0xFF] ^               \
             sh_U2[((s3)>>16)&0xFF] ^ sh_U3[((s2)>>24)&0xFF] ^ (rk)[1];\
        o2 = sh_U0[(s2)&0xFF] ^ sh_U1[((s1)>>8)&0xFF] ^               \
             sh_U2[((s0)>>16)&0xFF] ^ sh_U3[((s3)>>24)&0xFF] ^ (rk)[2];\
        o3 = sh_U0[(s3)&0xFF] ^ sh_U1[((s2)>>8)&0xFF] ^               \
             sh_U2[((s1)>>16)&0xFF] ^ sh_U3[((s0)>>24)&0xFF] ^ (rk)[3];\
    } while (0)

// ────────────────────────────────────────────────────────────────────────────────
// AES-256 ECB encryption kernel
__global__ void aes256_ecb_encrypt(const uint8_t* __restrict__ in,
                                   uint8_t* __restrict__ out,
                                   size_t nBlocks) {
    const size_t tid    = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t stride = blockDim.x * gridDim.x;
    const uint4* in4  = reinterpret_cast<const uint4*>(in);
    uint4* out4       = reinterpret_cast<uint4*>(out);
    const uint32_t* rk = d_roundKeys;

    __shared__ uint32_t sh_T0[256], sh_T1[256], sh_T2[256], sh_T3[256];
    __shared__ uint8_t  sh_sbox[256];
    if (threadIdx.x < 256) {
        sh_T0[threadIdx.x] = d_T0[threadIdx.x];
        sh_T1[threadIdx.x] = d_T1[threadIdx.x];
        sh_T2[threadIdx.x] = d_T2[threadIdx.x];
        sh_T3[threadIdx.x] = d_T3[threadIdx.x];
        sh_sbox[threadIdx.x] = d_sbox[threadIdx.x];
    }
    __syncthreads();

    for (size_t blk = tid * 2; blk < nBlocks; blk += stride * 2) {
        size_t blk2 = blk + 1;
        uint4 inBlock = in4[blk];
        uint32_t s0 = inBlock.x ^ rk[0];
        uint32_t s1 = inBlock.y ^ rk[1];
        uint32_t s2 = inBlock.z ^ rk[2];
        uint32_t s3 = inBlock.w ^ rk[3];

        uint32_t t0,t1,t2,t3;
#pragma unroll
        for (int r = 4; r <= 52; r += 4) {
            ENC_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk + r);
            s0=t0; s1=t1; s2=t2; s3=t3;
        }

        const uint8_t* sb = sh_sbox;
        uint32_t k0 = ((uint32_t)sb[s0 & 0xFF]) |
                      ((uint32_t)sb[s1 & 0xFF] << 8) |
                      ((uint32_t)sb[s2 & 0xFF] << 16) |
                      ((uint32_t)sb[s3 & 0xFF] << 24);
        uint32_t k1 = ((uint32_t)sb[(s1>>8)&0xFF]) |
                      ((uint32_t)sb[(s2>>8)&0xFF] << 8) |
                      ((uint32_t)sb[(s3>>8)&0xFF] << 16) |
                      ((uint32_t)sb[(s0>>8)&0xFF] << 24);
        uint32_t k2 = ((uint32_t)sb[(s2>>16)&0xFF]) |
                      ((uint32_t)sb[(s3>>16)&0xFF] << 8) |
                      ((uint32_t)sb[(s0>>16)&0xFF] << 16) |
                      ((uint32_t)sb[(s1>>16)&0xFF] << 24);
        uint32_t k3 = ((uint32_t)sb[(s3>>24)&0xFF]) |
                      ((uint32_t)sb[(s0>>24)&0xFF] << 8) |
                      ((uint32_t)sb[(s1>>24)&0xFF] << 16) |
                      ((uint32_t)sb[(s2>>24)&0xFF] << 24);

        k0 ^= rk[56]; k1 ^= rk[57]; k2 ^= rk[58]; k3 ^= rk[59];
        out4[blk] = make_uint4(k0,k1,k2,k3);

        if (blk2 < nBlocks) {
            inBlock = in4[blk2];
            s0 = inBlock.x ^ rk[0];
            s1 = inBlock.y ^ rk[1];
            s2 = inBlock.z ^ rk[2];
            s3 = inBlock.w ^ rk[3];
            for (int r = 4; r <= 52; r += 4) {
                ENC_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk + r);
                s0=t0; s1=t1; s2=t2; s3=t3;
            }
            k0 = ((uint32_t)sb[s0 & 0xFF]) |
                 ((uint32_t)sb[s1 & 0xFF] << 8) |
                 ((uint32_t)sb[s2 & 0xFF] << 16) |
                 ((uint32_t)sb[s3 & 0xFF] << 24);
            k1 = ((uint32_t)sb[(s1>>8)&0xFF]) |
                 ((uint32_t)sb[(s2>>8)&0xFF] << 8) |
                 ((uint32_t)sb[(s3>>8)&0xFF] << 16) |
                 ((uint32_t)sb[(s0>>8)&0xFF] << 24);
            k2 = ((uint32_t)sb[(s2>>16)&0xFF]) |
                 ((uint32_t)sb[(s3>>16)&0xFF] << 8) |
                 ((uint32_t)sb[(s0>>16)&0xFF] << 16) |
                 ((uint32_t)sb[(s1>>16)&0xFF] << 24);
            k3 = ((uint32_t)sb[(s3>>24)&0xFF]) |
                 ((uint32_t)sb[(s0>>24)&0xFF] << 8) |
                 ((uint32_t)sb[(s1>>24)&0xFF] << 16) |
                 ((uint32_t)sb[(s2>>24)&0xFF] << 24);
            k0 ^= rk[56]; k1 ^= rk[57]; k2 ^= rk[58]; k3 ^= rk[59];
            out4[blk2] = make_uint4(k0,k1,k2,k3);
        }
    }
}

// AES-256 ECB decryption kernel
__global__ void aes256_ecb_decrypt(const uint8_t* __restrict__ in,
                                   uint8_t* __restrict__ out,
                                   size_t nBlocks) {
    const size_t tid    = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t stride = blockDim.x * gridDim.x;
    const uint4* in4  = reinterpret_cast<const uint4*>(in);
    uint4* out4       = reinterpret_cast<uint4*>(out);
    const uint32_t* rk = d_roundKeys;

    __shared__ uint32_t sh_U0[256], sh_U1[256], sh_U2[256], sh_U3[256];
    __shared__ uint8_t  sh_isbox[256];
    if (threadIdx.x < 256) {
        sh_U0[threadIdx.x] = d_U0[threadIdx.x];
        sh_U1[threadIdx.x] = d_U1[threadIdx.x];
        sh_U2[threadIdx.x] = d_U2[threadIdx.x];
        sh_U3[threadIdx.x] = d_U3[threadIdx.x];
        sh_isbox[threadIdx.x] = d_inv_sbox[threadIdx.x];
    }
    __syncthreads();

    for (size_t blk = tid * 2; blk < nBlocks; blk += stride * 2) {
        size_t blk2 = blk + 1;
        uint4 inBlock = in4[blk];
        uint32_t s0 = inBlock.x ^ rk[56];
        uint32_t s1 = inBlock.y ^ rk[57];
        uint32_t s2 = inBlock.z ^ rk[58];
        uint32_t s3 = inBlock.w ^ rk[59];

        uint32_t t0,t1,t2,t3;
#pragma unroll
        for (int r = 52; r >= 4; r -= 4) {
            DEC_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk + r);
            s0=t0; s1=t1; s2=t2; s3=t3;
        }

        const uint8_t* isb = sh_isbox;
        uint32_t k0 = ((uint32_t)isb[s0 & 0xFF]) |
                      ((uint32_t)isb[s3 & 0xFF] << 8) |
                      ((uint32_t)isb[s2 & 0xFF] << 16) |
                      ((uint32_t)isb[s1 & 0xFF] << 24);
        uint32_t k1 = ((uint32_t)isb[(s1>>8)&0xFF]) |
                      ((uint32_t)isb[(s0>>8)&0xFF] << 8) |
                      ((uint32_t)isb[(s3>>8)&0xFF] << 16) |
                      ((uint32_t)isb[(s2>>8)&0xFF] << 24);
        uint32_t k2 = ((uint32_t)isb[(s2>>16)&0xFF]) |
                      ((uint32_t)isb[(s1>>16)&0xFF] << 8) |
                      ((uint32_t)isb[(s0>>16)&0xFF] << 16) |
                      ((uint32_t)isb[(s3>>16)&0xFF] << 24);
        uint32_t k3 = ((uint32_t)isb[(s3>>24)&0xFF]) |
                      ((uint32_t)isb[(s2>>24)&0xFF] << 8) |
                      ((uint32_t)isb[(s1>>24)&0xFF] << 16) |
                      ((uint32_t)isb[(s0>>24)&0xFF] << 24);

        k0 ^= rk[0]; k1 ^= rk[1]; k2 ^= rk[2]; k3 ^= rk[3];
        out4[blk] = make_uint4(k0,k1,k2,k3);

        if (blk2 < nBlocks) {
            inBlock = in4[blk2];
            s0 = inBlock.x ^ rk[56];
            s1 = inBlock.y ^ rk[57];
            s2 = inBlock.z ^ rk[58];
            s3 = inBlock.w ^ rk[59];
            for (int r = 52; r >= 4; r -= 4) {
                DEC_ROUND(t0,t1,t2,t3, s0,s1,s2,s3, rk + r);
                s0=t0; s1=t1; s2=t2; s3=t3;
            }
            k0 = ((uint32_t)isb[s0 & 0xFF]) |
                 ((uint32_t)isb[s3 & 0xFF] << 8) |
                 ((uint32_t)isb[s2 & 0xFF] << 16) |
                 ((uint32_t)isb[s1 & 0xFF] << 24);
            k1 = ((uint32_t)isb[(s1>>8)&0xFF]) |
                 ((uint32_t)isb[(s0>>8)&0xFF] << 8) |
                 ((uint32_t)isb[(s3>>8)&0xFF] << 16) |
                 ((uint32_t)isb[(s2>>8)&0xFF] << 24);
            k2 = ((uint32_t)isb[(s2>>16)&0xFF]) |
                 ((uint32_t)isb[(s1>>16)&0xFF] << 8) |
                 ((uint32_t)isb[(s0>>16)&0xFF] << 16) |
                 ((uint32_t)isb[(s3>>16)&0xFF] << 24);
            k3 = ((uint32_t)isb[(s3>>24)&0xFF]) |
                 ((uint32_t)isb[(s2>>24)&0xFF] << 8) |
                 ((uint32_t)isb[(s1>>24)&0xFF] << 16) |
                 ((uint32_t)isb[(s0>>24)&0xFF] << 24);
            k0 ^= rk[0]; k1 ^= rk[1]; k2 ^= rk[2]; k3 ^= rk[3];
            out4[blk2] = make_uint4(k0,k1,k2,k3);
        }
    }
}
